#include "hip/hip_runtime.h"
#include "AdeniumRender.cuh"

namespace adetrace {
void resetImage(float4 * pix, 
            uint n)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(n, 512);
    dim3 grid(nblk, 1, 1);
    
    resetImage_kernel<<< grid, block >>>(pix,
        n);
}

void setModelViewMatrix(float * src, uint size) 
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_modelViewMatrix), src, size);
}

void renderImage(float4 * pix,
                uint imageW,
                uint imageH,
                float fovWidth,
                float aspectRatio,
                int2 * nodes,
				Aabb * nodeAabbs,
				KeyValuePair * elementHash,
				int4 * elementVertices,
				float3 * elementPoints,
				int isOrthographic)
{
    uint nthread = 8;
    uint nblockX = iDivUp(imageW, nthread);
    uint nblockY = iDivUp(imageH, nthread);
    dim3 block(nthread, nthread, 1);
    dim3 grid(nblockX, nblockY, 1);
    if(isOrthographic)
        renderImage_kernel<64, 1> <<< grid, block, 16320 >>>(pix,
                        imageW, imageH,
                        fovWidth,
                        aspectRatio,
                        nodes,
                        nodeAabbs,
				elementHash,
				elementVertices,
				elementPoints);
	else
	    renderImage_kernel<64, 0> <<< grid, block, 16320 >>>(pix,
                        imageW, imageH,
                        fovWidth,
                        aspectRatio,
                        nodes,
                        nodeAabbs,
				elementHash,
				elementVertices,
				elementPoints);
}

}
