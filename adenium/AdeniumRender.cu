#include "hip/hip_runtime.h"
#include "AdeniumRender.cuh"

namespace adetrace {
void resetImage(float4 * pix, 
            uint n)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(n, 512);
    dim3 grid(nblk, 1, 1);
    
    resetImage_kernel<<< grid, block >>>(pix,
        n);
}

void setModelViewMatrix(float * src, uint size) 
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_modelViewMatrix), src, size);
}

void renderImageOrthographic(float4 * pix,
                uint imageW,
                uint imageH,
                float fovWidth,
                float aspectRatio)
{
    uint nthread = 8;
    uint nblockX = iDivUp(imageW, nthread);
    uint nblockY = iDivUp(imageH, nthread);
    dim3 block(nthread, nthread, 1);
    dim3 grid(nblockX, nblockY, 1);
    renderImageOrthographic_kernel<<< grid, block >>>(pix,
                        imageW, imageH,
                        fovWidth,
                        aspectRatio);
}

}
