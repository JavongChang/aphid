#include "hip/hip_runtime.h"
/*
 *  sort node count and range
 *
 *  1    0,13
 *  2    0,6  7,13
 *  4    0,3  4,6  7,10  11,13
 *  8    0,1  2,3  4,5   5,6   7,8  9,10  11,12  13,13
 *
 *  each node consumes one spawns two or one or zero
 *  level n needs max 2^n nodes
 *  each level has node count and range updated by atomic
 *  
 *  taskIn  1  0,13 |
 *  taskOut 3  0,13 | 0,6  7,13 
 *
 *  taskIn  3  0,13   0,6  7,13 |
 *  taskOut 7  0,13   0,6  7,13 | 0,3   4,6  7,10  11,13
 *
 *  taskIn  7  0,13   0,6  7,13   0,3   4,6  7,10  11,13 |
 *  taskOut 15 0,13   0,6  7,13   0,3   4,6  7,10  11,13 | 0,1  2,3  4,5   5,6   7,8  9,10  11,12  13,13
 *
 *  queue has   tbit where the first task is available
 *              qtail where the last task is available
 *  
 *  0  1  2  3 ... n-1
 *  ^              ^
 *  |              |
 *  tbid           qtail
 *
 *  dequeue() return task[tbid] tbid++
 *  only taskIn can be dequeued
 *  enqueue() set task[qtail] qtail++
 *  only taskOut can be enqueued
 *
 *  when taskIn.tbid > taskOut.qtail means no more work is available
 *  no more work is available doesn't mean the job is done
 *  still could be more task added, need workDoneCounter
 *  once workDoneCounter > taskOut.qtail, there is nothing left to do
 *  intially task out is a copy of task in
 *  new tasks will be added to end
 *  each time a task is done first enqueue() of needed, 
 *  then check if work is done, if not then dequeue()
 *  
 */
 
#include "quickSort.cuh"
#include "bvh_common.h"

extern "C" {
void cu_testQuickSort(void * q,
                    uint * idata,
                    uint * nodes, 
                    uint numNodes,
                    uint maxNumNodes,
                    uint maxNumParallelNodes,
                    uint * checkMaxN)
{
    printf("cu test quicksort\n");
    printf("init q max n works %i\n", maxNumNodes);
    simpleQueue::initSimpleQueue_kernel<<<512, 512>>>((simpleQueue::SimpleQueue *)q,
        numNodes -1,
        maxNumNodes);
    
    hipDeviceSynchronize();
    
    // SimpleQueue q(lock, tail, nodeRanges);
    const int tpb = 256;
    dim3 block(tpb, 1, 1);
// one warp per parallel node
    const unsigned nblk = maxNumParallelNodes>>5;
    dim3 grid(nblk, 1, 1);
    
    quickSort_checkQ_kernel<<<grid, block>>>(checkMaxN, (simpleQueue::SimpleQueue *)q);
}

}

