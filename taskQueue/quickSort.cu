#include "hip/hip_runtime.h"
/*
 *  sort node count and range
 *
 *  1    0,13
 *  2    0,6  7,13
 *  4    0,3  4,6  7,10  11,13
 *  8    0,1  2,3  4,5   5,6   7,8  9,10  11,12  13,13
 *
 *  each node consumes one spawns two or one or zero
 *  level n needs max 2^n nodes
 *  each level has node count and range updated by atomic
 *  
 *  taskIn  1  0,13 |
 *  taskOut 3  0,13 | 0,6  7,13 
 *
 *  taskIn  3  0,13   0,6  7,13 |
 *  taskOut 7  0,13   0,6  7,13 | 0,3   4,6  7,10  11,13
 *
 *  taskIn  7  0,13   0,6  7,13   0,3   4,6  7,10  11,13 |
 *  taskOut 15 0,13   0,6  7,13   0,3   4,6  7,10  11,13 | 0,1  2,3  4,5   5,6   7,8  9,10  11,12  13,13
 *
 *  when taskIn.tbid > taskOut.qtail means no more work is available
 *  no more work is available doesn't mean the job is done
 *  still could be more task added, need workDoneCounter
 *  once workDoneCounter > taskOut.qtail, there is nothing left to do
 *  intially task out is a copy of task in
 *  new tasks will be added to end
 *  each time a task is done first enqueue() of needed, 
 *  then check if work is done, if not then dequeue()
 *  
 */
 
#include "quickSort.cuh"
#include "bvh_common.h"

extern "C" {
void cu_testQuickSort(void * q,
                    uint * idata,
                    uint * nodes, 
                    SimpleQueueInterface * qi,
                    uint maxNumParallelNodes,
                    uint * checkMaxN)
{
    //hipDeviceSynchronize();
    const int tpb = 256;
    dim3 block(tpb, 1, 1);
// one warp per parallel node
    const unsigned nblk = maxNumParallelNodes;
    dim3 grid(nblk, 1, 1);
    
    quickSort_checkQ_kernel<<<grid, block>>>(checkMaxN, 
                                (simpleQueue::SimpleQueue *)q,
                                qi,
                                idata,
                                (int2 *)nodes);
}

}

