#include "hip/hip_runtime.h"
#include <bvh_common.h>
#include "bvh_math.cuh"
__global__ void computeMass_kernel(float * dst,
                float * mass0,
                uint * anchored,
                float scale,
                uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	float m0 = mass0[ind];
	if(anchored[ind] == 0) dst[ind] = m0 * scale;
}

__global__ void useAllAnchoredVelocity_kernel(float3 * vel,
                                float3 * anchoredVel,
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    float3 va = anchoredVel[ind];
    vel[ind] = va;
}

__global__ void useAnchoredVelocity_kernel(float3 * vel,
                                float3 * anchoredVel,
								uint * anchored,
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    float3 va = anchoredVel[ind];
    if(anchored[ind] > 0) vel[ind] = va;
}

__global__ void integrate_kernel(float3 * pos, 
								float3 * vel,
                                float3 * vela,
								float dt, 
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    float3 anchoredVel = vela[ind];
    vel[ind] = anchoredVel;
	float3_add_inplace(pos[ind], scale_float3_by(anchoredVel, dt));
}

__global__ void integrate1_kernel(float3 * pos, 
								float3 * vel,
                                float3 * anchoredVel,
								uint * anchor,
								float dt, 
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    float3 va = anchoredVel[ind];
    if(anchor[ind] > 0) vel[ind] = va;
	else va = vel[ind];
	float3_add_inplace(pos[ind], scale_float3_by(va, dt));
}

__global__ void integrate2_kernel(float3 * pos, 
								float3 * vel,
                                float dt, 
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    float3 va = vel[ind];
	float3_add_inplace(pos[ind], scale_float3_by(va, dt));
}

__global__ void addGravity_kernel(float3 * deltaVel, 
								uint * anchored,
                                float dt, 
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    if(anchored[ind] > 0) deltaVel[ind] = make_float3(0.f, 0.f, 0.f);
    else deltaVel[ind] = make_float3(0.f, -0.1635f, 0.f);
}

__global__ void impulseForce_kernel(float3 * force,
                           float3 * deltaVel,
                           float * mass,
                           float dt,
                           uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
/*
 *   F = J / dt
 *   J = m * dv
 */
    float m = mass[ind];
    if(m > 1e5f) force[ind] = make_float3(0.f, 0.f, 0.f);
    else force[ind] = scale_float3_by(deltaVel[ind], m / dt);
}

namespace masssystem {
void computeMass(float * dst,
                float * mass0,
                uint * anchored,
                float scale,
                uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    computeMass_kernel<<< grid, block >>>(dst,
        mass0,
        anchored,
        scale,
        maxInd);
}

void useAnchoredVelocity(float3 * vel, 
                float3 * anchoredVel,
                uint * anchored,
                uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    useAnchoredVelocity_kernel<<< grid, block >>>(vel,
        anchoredVel,
        anchored,
        maxInd);
}

void useAllAnchoredVelocity(float3 * vel, 
                float3 * anchoredVel,
                uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    useAllAnchoredVelocity_kernel<<< grid, block >>>(vel,
        anchoredVel,
        maxInd);
}

void integrate(float3 * pos, 
								float3 * vel, 
                                float3 * anchoredVel,
								uint * anchor,
								float dt, 
								uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    integrate1_kernel<<< grid, block >>>(pos,
        vel,
        anchoredVel,
        anchor,
        dt,
        maxInd);
}

void integrateAllAnchored(float3 * pos,
                    float3 * vel,
                    float3 * vela,
                    float dt,
                    uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    integrate_kernel<<< grid, block >>>(pos,
        vel,
        vela,
        dt,
        maxInd);
}

void integrateSimple(float3 * pos, 
                float3 * vel, 
                float dt, 
                uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    integrate2_kernel<<< grid, block >>>(pos,
        vel,
        dt,
        maxInd);
}

void addGravity(float3 * deltaVel,
                uint * anchored,
                float dt,
                uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    addGravity_kernel<<< grid, block >>>(deltaVel,
        anchored,
        dt,
        maxInd);
}

void impulseForce(float3 * force,
                           float3 * deltaVel,
                           float * mass,
                           float dt,
                           uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    impulseForce_kernel<<< grid, block >>>(force,
                                           deltaVel,
        mass,
        dt,
        maxInd);
}

}
