#include "hip/hip_runtime.h"
#include <bvh_common.h>
#include "bvh_math.cuh"
__global__ void computeMass_kernel(float * dst,
                float * mass0,
                uint * anchored,
                float scale,
                uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	float m0 = mass0[ind];
	if(anchored[ind] == 0) dst[ind] = m0 * scale;
}

__global__ void integrate_kernel(float3 * pos, 
								float3 * vel,
                                float3 * vela,
								float dt, 
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    float3 anchoredVel = vela[ind];
    vel[ind] = anchoredVel;
	float3_add_inplace(pos[ind], scale_float3_by(anchoredVel, dt));
}

__global__ void integrate_kernel1(float3 * pos, 
								float3 * vel,
                                float3 * anchoredVel,
								uint * anchor,
								float dt, 
								uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
    float3 va = anchoredVel[ind];
    if(anchor[ind] > 0) vel[ind] = va;
	else va = vel[ind];
	float3_add_inplace(pos[ind], scale_float3_by(va, dt));
}

namespace masssystem {
void computeMass(float * dst,
                float * mass0,
                uint * anchored,
                float scale,
                uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    computeMass_kernel<<< grid, block >>>(dst,
        mass0,
        anchored,
        scale,
        maxInd);
}

void integrate(float3 * pos, 
								float3 * vel, 
                                float3 * anchoredVel,
								uint * anchor,
								float dt, 
								uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    integrate_kernel1<<< grid, block >>>(pos,
        vel,
        anchoredVel,
        anchor,
        dt,
        maxInd);
}

void integrateAllAnchored(float3 * pos,
                    float3 * vel,
                    float3 * vela,
                    float dt,
                    uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    integrate_kernel<<< grid, block >>>(pos,
        vel,
        vela,
        dt,
        maxInd);
}

}
