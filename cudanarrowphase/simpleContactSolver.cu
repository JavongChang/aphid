#include "hip/hip_runtime.h"
#include "simpleContactSolver_implement.h"
#include <bvh_math.cu>

inline __device__ uint4 computePointIndex(uint * pointStarts,
                                            uint * indexStarts,
                                            uint4 * indices,
                                            uint combined)
{
    const uint objI = extractObjectInd(combined);
    const uint elmI = extractElementInd(combined);
    
    uint4 r;
    r.x = pointStarts[objI] + indices[indexStarts[objI] + elmI].x;
    r.y = pointStarts[objI] + indices[indexStarts[objI] + elmI].y;
    r.z = pointStarts[objI] + indices[indexStarts[objI] + elmI].z;
    r.w = pointStarts[objI] + indices[indexStarts[objI] + elmI].w;
    return r;
}

__global__ void writeContactIndex_kernel(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufferLength) return;
	
	if(ind < n) {
	    dstInd[ind].key = srcInd[ind];
	    dstInd[ind].value = ind >> 1;
	}
	else {
	    dstInd[ind].key = 1e30;
	    dstInd[ind].value = 1e30;
	}
}

__global__ void computeSplitBufLoc_kernel(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufLength) return;
	
	const uint dstLoc = bodyPairHash[ind].value;
	if(srcPairs[dstLoc].x == bodyPairHash[ind].key) {
	    splits[dstLoc].x = ind;
	}
	else {
	    splits[dstLoc].y = ind;
	}
}

__global__ void countUniqueBody_kernel(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint num, uint bufLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufLength) return;

	if(ind >= num) {
	    dstCount[ind] = 0;
	    return;
	}
	
	dstCount[ind] = 0;
	
	const uint a = srcInd[ind].key;
	
	int isFirst = 0;
	
	if(ind < 1) isFirst = 1;
	else if(srcInd[ind - 1].key != a) isFirst = 1;
	
	if(!isFirst) return;
	
	dstCount[ind] = 1;

	unsigned cur = ind;
// check backward
	for(;;) {
	    if(cur == num - 1) return;
	    cur++;
	    if(srcInd[cur].key != a) return;
	    dstCount[ind]++;
	}	
}

__global__ void stopAtContact_kernel(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * pointStarts,
                        uint * indexStarts,
                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].x);
	const uint4 ib = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].y);
	
	dstVelocity[ia.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.w] = make_float3(0.f, 0.f, 0.f);
	
	dstVelocity[ib.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.w] = make_float3(0.f, 0.f, 0.f);
}

extern "C" {
    
void simpleContactSolverWriteContactIndex(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufferLength, 512);
    dim3 grid(nblk, 1, 1);
    
    writeContactIndex_kernel<<< grid, block >>>(dstInd, 
                                                srcInd,
                                                n, bufferLength);
}

void simpleContactSolverComputeSplitBufLoc(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitBufLoc_kernel<<< grid, block >>>(splits, 
                                        srcPairs, 
                                        bodyPairHash, 
                                        bufLength);
}

void simpleContactSolverCountUniqueBody(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint num, uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    countUniqueBody_kernel<<< grid, block >>>(dstCount,
                                     srcInd, 
                                       num, bufLength);
}

void simpleContactSolverStopAtContact(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * objectPointStarts,
                        uint * objectIndexStarts,
                        uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    stopAtContact_kernel<<< grid, block >>>(dstVelocity, 
                        pairs,
                        indices,
                        objectPointStarts,
                        objectIndexStarts,
                        numContacts);
}

}
