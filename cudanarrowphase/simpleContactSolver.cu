#include "hip/hip_runtime.h"
#include "simpleContactSolver_implement.h"
#include <bvh_math.cu>
#include <CudaBase.h>

inline __device__ uint4 computePointIndex(uint * pointStarts,
                                            uint * indexStarts,
                                            uint4 * indices,
                                            uint combined)
{
    const uint objI = extractObjectInd(combined);
    const uint elmI = extractElementInd(combined);
    
    uint4 r;
    r.x = pointStarts[objI] + indices[indexStarts[objI] + elmI].x;
    r.y = pointStarts[objI] + indices[indexStarts[objI] + elmI].y;
    r.z = pointStarts[objI] + indices[indexStarts[objI] + elmI].z;
    r.w = pointStarts[objI] + indices[indexStarts[objI] + elmI].w;
    return r;
}

inline __device__ void computeBodyAngularVelocity(float3 & angularVel,
                                                  float3 averageLinearVel,
                                                  float3 * position,
                                                  float3 * velocity,
                                                  uint4 ind)
{
    float3 center;
	float3_average4(center, position, ind);
	
	float3 omega[4];
// omega = r cross v
// v = omega cross r
    omega[0] = float3_cross(float3_difference(position[ind.x], center), float3_difference(velocity[ind.x], averageLinearVel));
    omega[1] = float3_cross(float3_difference(position[ind.y], center), float3_difference(velocity[ind.y], averageLinearVel));
    omega[2] = float3_cross(float3_difference(position[ind.z], center), float3_difference(velocity[ind.z], averageLinearVel));
    omega[3] = float3_cross(float3_difference(position[ind.w], center), float3_difference(velocity[ind.w], averageLinearVel));
    
	float3_average4_direct(angularVel, omega);
}

inline __device__ void computeBodyVelocities1(uint * pointStarts, 
                                                uint * indexStarts, 
                                                uint4 * indices, 
                                                uint ind,
                                                float3 * position,
                                                float3 * velocity, 
                                                float3 & linearVelocity, 
                                                float3 & angularVelocity)
{
    const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, ind);
	
	float3_average4(linearVelocity, velocity, ia);
	
	computeBodyAngularVelocity(angularVelocity, linearVelocity, position, velocity, ia);
}

inline __device__ void computeBodyVelocities(uint * pointStarts, 
                                                uint * indexStarts, 
                                                uint4 * indices, 
                                                uint2 pair,
                                                float3 * position,
                                                float3 * velocity, 
                                                float3 & linearVelocityA, 
                                                float3 & linearVelocityB,
                                                float3 & angularVelocityA, 
                                                float3 & angularVelocityB)
{
    const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, pair.x);
	const uint4 ib = computePointIndex(pointStarts, indexStarts, indices, pair.y);
	
	float3_average4(linearVelocityA, velocity, ia);
	float3_average4(linearVelocityB, velocity, ib);
	
	float3 centerA;
	float3_average4(centerA, position, ia);
	
	float3 centerB;
	float3_average4(centerB, position, ib);
}

inline __device__ uint getBodyCountAt(uint ind, uint * count)
{
    uint cur = ind;
    for(;;) {
        if(count[ind] > 0) return count[ind];
        cur--;
    }
}

// pointing inside A
inline __device__ float3 normalOnA(const ContactData & contact)
{
    return float3_normalize(float3_from_float4(contact.separateAxis));
}

inline __device__ float computeRelativeVelocity(float3 nA,
                            float3 nB,
                            float3 linearVelocityA, 
                            float3 linearVelocityB,
                            float3 torqueA,
                            float3 torqueB,
                            float3 angularVelocityA, 
                            float3 angularVelocityB)
{
    return float3_dot(linearVelocityA, nA) +
            float3_dot(linearVelocityB, nB) +
            float3_dot(torqueA, angularVelocityA) +
            float3_dot(torqueB, angularVelocityB);
}

inline __device__ float computeDeltaLambda(float & accumulated, float lambda)
{
    const float last = accumulated;
    accumulated += lambda;
    if(accumulated < 0.f) accumulated = 0.f;
	return accumulated - last;
}

inline __device__ void computeDeltaVelocity(float3 & dst, float J, float3 N)
{
    dst = float3_add(dst, scale_float3_by(N, J));
}

inline __device__ float computeMassTensor(float3 nA, float3 nB, 
                                        float3 rA, float3 rB,
                                        float invMassA, float invMassB)
{
    float3 torqueA = float3_cross(rA, nA);
    float3 torqueB = float3_cross(rB, nB);
    float3 jmjA = float3_cross( scale_float3_by(torqueA, invMassA), rA );
    float3 jmjB = float3_cross( scale_float3_by(torqueB, invMassB), rB );
    
    return -1.f/(invMassA + invMassB + 
        float3_dot(jmjA, nA) + 
        float3_dot(jmjB, nB));
}

__global__ void writeContactIndex_kernel(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufferLength) return;
	
	if(ind < n) {
	    dstInd[ind].key = srcInd[ind];
	    dstInd[ind].value = ind >> 1;
	}
	else {
	    dstInd[ind].key = 1<<30;
	    dstInd[ind].value = 1<<30;
	}
}

__global__ void computeSplitBufLoc_kernel(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufLength) return;
	
	const uint dstLoc = bodyPairHash[ind].value;
	if(srcPairs[dstLoc].x == bodyPairHash[ind].key) {
	    splits[dstLoc].x = ind;
	}
	else {
	    splits[dstLoc].y = ind;
	}
}

__global__ void countBody_kernel(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dstCount[ind] = 0;
	
	const uint a = srcInd[ind].key;
	
	int isFirst = 0;
	
	if(ind < 1) isFirst = 1;
	else if(srcInd[ind - 1].key != a) isFirst = 1;
	
	if(!isFirst) return;
	
	dstCount[ind] = 1;

	unsigned cur = ind;
// check backward
	for(;;) {
	    if(cur == maxInd - 1) return;
	    cur++;
	    if(srcInd[cur].key != a) return;
	    dstCount[ind]++;
	}	
}

__global__ void computeSplitInvMass_kernel(float * invMass, 
                                        uint * bodyCount, 
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	uint n = getBodyCountAt(ind, bodyCount);
	
	invMass[ind] = 1.f * (float)n;
}

__global__ void setContactConstraint_kernel(float3 * projLinVel,
                                        float3 * projAngVel,
                                        float * lambda,
                                        float * Minv,
                                        uint2 * splits,
                                        uint2 * pairs,
                                        float3 * srcPos,
                                        float3 * srcVel,
                                        uint4 * indices,
                                        uint * pointStarts,
                                        uint * indexStarts,
                                        float * splitMass,
                                        ContactData * contacts,
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	lambda[ind] = 0.f;
	
	const uint2 dstInd = splits[ind];
	
	computeBodyVelocities1(pointStarts, indexStarts, indices, pairs[ind].x, srcPos, srcVel, 
	    projLinVel[dstInd.x], projAngVel[dstInd.x]);
	
	computeBodyVelocities1(pointStarts, indexStarts, indices, pairs[ind].y, srcPos, srcVel, 
	    projLinVel[dstInd.y], projAngVel[dstInd.y]);
	
	ContactData contact = contacts[ind];
	float3 nA = normalOnA(contact);
	float3 nB = float3_reverse(nA);
	
	Minv[ind] = computeMassTensor(nA, nB, contact.localA, contact.localB,
	                            splitMass[dstInd.x], splitMass[dstInd.y]);
}

__global__ void clearDeltaVelocity_kernel(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	deltaLinVel[ind] = make_float3(0.f, 0.f, 0.f);
	deltaAngVel[ind] = make_float3(0.f, 0.f, 0.f);
}

__global__ void stopAtContact_kernel(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * pointStarts,
                        uint * indexStarts,
                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].x);
	const uint4 ib = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].y);
	
	dstVelocity[ia.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.w] = make_float3(0.f, 0.f, 0.f);
	
	dstVelocity[ib.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.w] = make_float3(0.f, 0.f, 0.f);
}

__global__ void solveContact_kernel(float * lambda,
                        float3 * linearVelocity,
	                    float3 * angularVelocity,
	                    uint2 * splits,
	                    float * splitMass,
	                    float * Minv,
                        ContactData * contacts,
                        uint maxInd,
                        float * deltaJ,
                        float3 * relV,
                        int it)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint2 dstInd = splits[ind];
	
	relV[ind * JACOBI_NUM_ITERATIONS + it] = angularVelocity[dstInd.x];
	
	ContactData contact = contacts[ind];

	float3 nA = normalOnA(contact);
	float3 nB = float3_reverse(nA);
// N pointing inside object
// T = r X N	
	float3 torqueA = float3_cross(contact.localA, nA);
	float3 torqueB = float3_cross(contact.localB, nB);
	
	float J = computeRelativeVelocity(nA, nB,
	                        linearVelocity[dstInd.x], linearVelocity[dstInd.y],
	                        torqueA, torqueB,
	                        angularVelocity[dstInd.x], angularVelocity[dstInd.y]);
	
	J *= Minv[ind];
	
	const float invMassA = splitMass[dstInd.x];
	const float invMassB = splitMass[dstInd.y];
	
	float dJ = computeDeltaLambda(lambda[ind], J);
	
	computeDeltaVelocity(linearVelocity[dstInd.x], dJ * invMassA, nA);
	computeDeltaVelocity(linearVelocity[dstInd.y], dJ * invMassB, nB);
	
	computeDeltaVelocity(angularVelocity[dstInd.x], dJ * invMassA, torqueA);
	computeDeltaVelocity(angularVelocity[dstInd.y], dJ * invMassB, torqueB);
	
	deltaJ[ind * JACOBI_NUM_ITERATIONS + it] = dJ;
}

extern "C" {
    
void simpleContactSolverWriteContactIndex(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufferLength, 512);
    dim3 grid(nblk, 1, 1);
    
    writeContactIndex_kernel<<< grid, block >>>(dstInd, 
                                                srcInd,
                                                n, bufferLength);
}

void simpleContactSolverComputeSplitBufLoc(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitBufLoc_kernel<<< grid, block >>>(splits, 
                                        srcPairs, 
                                        bodyPairHash, 
                                        bufLength);
}

void simpleContactSolverCountBody(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint num)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(num, 512);
    dim3 grid(nblk, 1, 1);
    
    countBody_kernel<<< grid, block >>>(dstCount,
                                     srcInd, 
                                       num);
}

void simpleContactSolverComputeSplitInverseMass(float * invMass, 
                                        uint * bodyCount, 
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitInvMass_kernel<<< grid, block >>>(invMass,
                                     bodyCount, 
                                       bufLength);
}

void simpleContactSolverSetContactConstraint(float3 * projLinVel,
                                        float3 * projAngVel,
                                        float * lambda,
                                        float * Minv,
                                        uint2 * splits,
                                        uint2 * pairs,
                                        float3 * pos,
                                        float3 * vel,
                                        uint4 * ind,
                                        uint * perObjPointStart,
                                        uint * perObjectIndexStart,
                                        float * splitMass,
                                        ContactData * contacts,
                                        uint numContacts)
{
    uint tpb = CudaBase::LimitNThreadPerBlock(32, 56);

    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(numContacts, tpb);
    dim3 grid(nblk, 1, 1);
    
    setContactConstraint_kernel<<< grid, block >>>(projLinVel,
                                        projAngVel,
                                        lambda,
                                        Minv,
                                        splits,
                                        pairs,
                                        pos,
                                        vel,
                                        ind,
                                        perObjPointStart,
                                        perObjectIndexStart,
                                        splitMass,
                                        contacts,
                                        numContacts);
}

void simpleContactSolverClearDeltaVelocity(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    clearDeltaVelocity_kernel<<< grid, block >>>(deltaLinVel,
                                     deltaAngVel, 
                                       bufLength);
}

void simpleContactSolverStopAtContact(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * objectPointStarts,
                        uint * objectIndexStarts,
                        uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    stopAtContact_kernel<<< grid, block >>>(dstVelocity, 
                        pairs,
                        indices,
                        objectPointStarts,
                        objectIndexStarts,
                        numContacts);
}

void simpleContactSolverSolveContact(float * lambda,
                        float3 * linearVelocity,
	                    float3 * angularVelocity,
	                    uint2 * splits,
	                    float * splitMass,
	                    float * Minv,
                        ContactData * contacts,
                        uint numContacts,
                        float * deltaJ,
                        float3 * relV,
                        int it)
{
    uint tpb = CudaBase::LimitNThreadPerBlock(24, 40);

    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(numContacts, tpb);
    dim3 grid(nblk, 1, 1);
    
    solveContact_kernel<<< grid, block >>>(lambda,
                        linearVelocity,
	                    angularVelocity,
	                    splits,
	                    splitMass,
	                    Minv,
                        contacts,
                        numContacts,
                        deltaJ,
                        relV,
                        it);
}

}
