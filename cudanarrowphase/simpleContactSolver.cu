#include "hip/hip_runtime.h"
#include "simpleContactSolver_implement.h"
#include <bvh_math.cu>

inline __device__ uint4 computePointIndex(uint * pointStarts,
                                            uint * indexStarts,
                                            uint4 * indices,
                                            uint combined)
{
    const uint objI = extractObjectInd(combined);
    const uint elmI = extractElementInd(combined);
    
    uint4 r;
    r.x = pointStarts[objI] + indices[indexStarts[objI] + elmI].x;
    r.y = pointStarts[objI] + indices[indexStarts[objI] + elmI].y;
    r.z = pointStarts[objI] + indices[indexStarts[objI] + elmI].z;
    r.w = pointStarts[objI] + indices[indexStarts[objI] + elmI].w;
    return r;
}

inline __device__ void computeBodyAngularVelocity(float3 & angularVel,
                                                  float3 averageLinearVel,
                                                  float3 * position,
                                                  float3 * velocity,
                                                  uint4 ind)
{
    float3 center;
	float3_average4(center, position, ind);
	
	float3 omega[4];
// omega = r cross v
// v = omega cross r
    float3_cross1(omega[0], float3_difference(position[ind.x], center), float3_difference(velocity[ind.x], averageLinearVel));
    float3_cross1(omega[1], float3_difference(position[ind.y], center), float3_difference(velocity[ind.y], averageLinearVel));
    float3_cross1(omega[2], float3_difference(position[ind.z], center), float3_difference(velocity[ind.z], averageLinearVel));
    float3_cross1(omega[3], float3_difference(position[ind.w], center), float3_difference(velocity[ind.w], averageLinearVel));
    
	float3_average4_direct(angularVel, omega);
}

inline __device__ void computeBodyVelocities1(uint * pointStarts, 
                                                uint * indexStarts, 
                                                uint4 * indices, 
                                                uint ind,
                                                float3 * position,
                                                float3 * velocity, 
                                                float3 & linearVelocityA, 
                                                float3 & angularVelocityA)
{
    const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, ind);
	
	float3_average4(linearVelocityA, velocity, ia);

	computeBodyAngularVelocity(angularVelocityA, linearVelocityA, position, velocity, ia);
}

inline __device__ void computeBodyVelocities(uint * pointStarts, 
                                                uint * indexStarts, 
                                                uint4 * indices, 
                                                uint2 pair,
                                                float3 * position,
                                                float3 * velocity, 
                                                float3 & linearVelocityA, 
                                                float3 & linearVelocityB,
                                                float3 & angularVelocityA, 
                                                float3 & angularVelocityB)
{
    const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, pair.x);
	const uint4 ib = computePointIndex(pointStarts, indexStarts, indices, pair.y);
	
	float3_average4(linearVelocityA, velocity, ia);
	float3_average4(linearVelocityB, velocity, ib);
	
	float3 centerA;
	float3_average4(centerA, position, ia);
	
	float3 centerB;
	float3_average4(centerB, position, ib);
	
	
}

inline __device__ uint getBodyCountAt(uint ind, uint * count)
{
    uint cur = ind;
    for(;;) {
        if(count[ind] > 0) return count[ind];
        cur--;
    }
}

inline __device__ void 	collide(float3 linearVelocityA, 
                            float3 linearVelocityB,
                            float massA, 
                            float massB,
                            float3 & deltaLinVelA,
                            float3 & deltaLinVelB)
{
    
}

__global__ void writeContactIndex_kernel(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufferLength) return;
	
	if(ind < n) {
	    dstInd[ind].key = srcInd[ind];
	    dstInd[ind].value = ind >> 1;
	}
	else {
	    dstInd[ind].key = 1<<30;
	    dstInd[ind].value = 1<<30;
	}
}

__global__ void computeSplitBufLoc_kernel(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufLength) return;
	
	const uint dstLoc = bodyPairHash[ind].value;
	if(srcPairs[dstLoc].x == bodyPairHash[ind].key) {
	    splits[dstLoc].x = ind;
	}
	else {
	    splits[dstLoc].y = ind;
	}
}

__global__ void countBody_kernel(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dstCount[ind] = 0;
	
	const uint a = srcInd[ind].key;
	
	int isFirst = 0;
	
	if(ind < 1) isFirst = 1;
	else if(srcInd[ind - 1].key != a) isFirst = 1;
	
	if(!isFirst) return;
	
	dstCount[ind] = 1;

	unsigned cur = ind;
// check backward
	for(;;) {
	    if(cur == maxInd - 1) return;
	    cur++;
	    if(srcInd[cur].key != a) return;
	    dstCount[ind]++;
	}	
}

__global__ void computeSplitInvMass_kernel(float * invMass, 
                                        uint * bodyCount, 
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	uint n = getBodyCountAt(ind, bodyCount);
	
	invMass[ind] = 1.f * (float)n;
}

__global__ void setContactConstraint_kernel(float3 * linVelA,
                                             float3 * linVelB,
                                        float3 * angVelA,
                                        float3 * angVelB,
                                        float * lambda,
                                        float * invMass, 
                                        float * splitInvMass, 
                                        uint2 * splits, 
                                        uint2 * pairs,
                                        float3 * srcPos,
                                        float3 * srcVel,
                                        uint4 * indices,
                                        uint * pointStarts,
                                        uint * indexStarts,
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint2 massInd = splits[ind];
	
	float invMassA = splitInvMass[massInd.x];
	float invMassB = splitInvMass[massInd.y];
	
	invMass[ind] = 1.f / (invMassA + invMassB);
	lambda[ind] = 0.f;
	
	float3 llinVelA, langVelA;
	computeBodyVelocities1(pointStarts, indexStarts, indices, pairs[ind].x, srcPos, srcVel, 
	    llinVelA, langVelA);
	
	float3 llinVelB, langVelB;
	computeBodyVelocities1(pointStarts, indexStarts, indices, pairs[ind].y, srcPos, srcVel, 
	    llinVelB, langVelB);
	
	angVelA[ind] = langVelA;
	angVelB[ind] = langVelB;
	linVelA[ind] = llinVelA;
	linVelB[ind] = llinVelB;
}

__global__ void clearDeltaVelocity_kernel(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	deltaLinVel[ind] = make_float3(0.f, 0.f, 0.f);
	deltaAngVel[ind] = make_float3(0.f, 0.f, 0.f);
}

__global__ void stopAtContact_kernel(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * pointStarts,
                        uint * indexStarts,
                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].x);
	const uint4 ib = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].y);
	
	dstVelocity[ia.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.w] = make_float3(0.f, 0.f, 0.f);
	
	dstVelocity[ib.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.w] = make_float3(0.f, 0.f, 0.f);
}

__global__ void solveContact_kernel(float3 * deltaLinVel,
                        float3 * deltaAngVel,
	                    uint2 * splits,
	                    float * splitMass,
	                    float3 * srcVelocity,
                    uint2 * pairs,
                    uint4 * indices,
                    uint * pointStarts,
                    uint * indexStarts,
                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	float3 linearVelocityA, linearVelocityB, angularVelocityA, angularVelocityB;
	computeBodyVelocities(pointStarts, indexStarts, indices, pairs[ind], srcVelocity, srcVelocity,
	    linearVelocityA, linearVelocityB, angularVelocityA, angularVelocityB);
	
	const uint2 dstInd = splits[ind];
	float massA = splitMass[dstInd.x];
	float massB = splitMass[dstInd.y];
	
	collide(linearVelocityA, linearVelocityB,
	        massA, massB,
	        deltaLinVel[dstInd.x],
	        deltaLinVel[dstInd.y]
	        );
}

extern "C" {
    
void simpleContactSolverWriteContactIndex(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufferLength, 512);
    dim3 grid(nblk, 1, 1);
    
    writeContactIndex_kernel<<< grid, block >>>(dstInd, 
                                                srcInd,
                                                n, bufferLength);
}

void simpleContactSolverComputeSplitBufLoc(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitBufLoc_kernel<<< grid, block >>>(splits, 
                                        srcPairs, 
                                        bodyPairHash, 
                                        bufLength);
}

void simpleContactSolverCountBody(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint num)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(num, 512);
    dim3 grid(nblk, 1, 1);
    
    countBody_kernel<<< grid, block >>>(dstCount,
                                     srcInd, 
                                       num);
}

void simpleContactSolverComputeSplitInverseMass(float * invMass, 
                                        uint * bodyCount, 
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitInvMass_kernel<<< grid, block >>>(invMass,
                                     bodyCount, 
                                       bufLength);
}

void simpleContactSolverSetContactConstraint(float3 * linVelA,
                                             float3 * linVelB,
                                        float3 * angVelA,
                                        float3 * angVelB,
                                        float * lambda,
                                        float * invMass, 
                                        float * splitInvMass, 
                                        uint2 * splits, 
                                        uint2 * pairs,
                                        float3 * pos,
                                        float3 * vel,
                                        uint4 * ind,
                                        uint * perObjPointStart,
                                        uint * perObjectIndexStart,
                                        uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    setContactConstraint_kernel<<< grid, block >>>(linVelA,
                                        linVelB,
                                        angVelA,
                                        angVelB,
                                        lambda,
                                        invMass, 
                                        splitInvMass, 
                                        splits, 
                                        pairs,
                                        pos,
                                        vel,
                                        ind,
                                        perObjPointStart,
                                        perObjectIndexStart,
                                        numContacts);
}

void simpleContactSolverClearDeltaVelocity(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    clearDeltaVelocity_kernel<<< grid, block >>>(deltaLinVel,
                                     deltaAngVel, 
                                       bufLength);
}

void simpleContactSolverStopAtContact(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * objectPointStarts,
                        uint * objectIndexStarts,
                        uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    stopAtContact_kernel<<< grid, block >>>(dstVelocity, 
                        pairs,
                        indices,
                        objectPointStarts,
                        objectIndexStarts,
                        numContacts);
}

void simpleContactSolverSolveContact(float3 * deltaLinVel,
	                    float3 * deltaAngVel,
	                    uint2 * splits,
	                    float * splitMass,
	                    float3 * srcVelocity,
                    uint2 * pairs,
                    uint4 * indices,
                    uint * objectPointStarts,
                    uint * objectIndexStarts,
                    uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    solveContact_kernel<<< grid, block >>>(deltaLinVel,
	                    deltaAngVel,
	                    splits,
	                    splitMass,
	                    srcVelocity,
                        pairs,
                        indices,
                        objectPointStarts,
                        objectIndexStarts,
                        numContacts);
}

}
