#include "hip/hip_runtime.h"
#include "simpleContactSolver_implement.h"
#include <bvh_math.cu>

inline __device__ uint4 computePointIndex(uint * pointStarts,
                                            uint * indexStarts,
                                            uint4 * indices,
                                            uint combined)
{
    const uint objI = extractObjectInd(combined);
    const uint elmI = extractElementInd(combined);
    
    uint4 r;
    r.x = pointStarts[objI] + indices[indexStarts[objI] + elmI].x;
    r.y = pointStarts[objI] + indices[indexStarts[objI] + elmI].y;
    r.z = pointStarts[objI] + indices[indexStarts[objI] + elmI].z;
    r.w = pointStarts[objI] + indices[indexStarts[objI] + elmI].w;
    return r;
}

inline __device__ void computeBodyLinearVelocity(uint * pointStarts, 
                                                uint * indexStarts, 
                                                uint4 * indices, 
                                                uint2 pair, 
                                                float3 * velocity, 
                                                float3 & linearVelocityA, 
                                                float3 & linearVelocityB)
{
    const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, pair.x);
	const uint4 ib = computePointIndex(pointStarts, indexStarts, indices, pair.y);
	
	linearVelocityA = velocity[ia.x];
	linearVelocityA = float3_add(linearVelocityA, velocity[ia.y]);
	linearVelocityA = float3_add(linearVelocityA, velocity[ia.z]);
	linearVelocityA = float3_add(linearVelocityA, velocity[ia.w]);
	linearVelocityA = scale_float3_by(linearVelocityA, 0.25f);
	
	linearVelocityB = velocity[ib.x];
	linearVelocityB = float3_add(linearVelocityA, velocity[ib.y]);
	linearVelocityB = float3_add(linearVelocityA, velocity[ib.z]);
	linearVelocityB = float3_add(linearVelocityA, velocity[ib.w]);
	linearVelocityB = scale_float3_by(linearVelocityB, 0.25f);
}

inline __device__ uint getBodyCountAt(uint ind, uint * count)
{
    uint cur = ind;
    for(;;) {
        if(count[ind] > 0) return count[ind];
        cur--;
    }
}

inline __device__ void 	collide(float3 linearVelocityA, 
                            float3 linearVelocityB,
                            float massA, 
                            float massB,
                            float3 & deltaLinVelA,
                            float3 & deltaLinVelB)
{
    
}

__global__ void writeContactIndex_kernel(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufferLength) return;
	
	if(ind < n) {
	    dstInd[ind].key = srcInd[ind];
	    dstInd[ind].value = ind >> 1;
	}
	else {
	    dstInd[ind].key = 1<<30;
	    dstInd[ind].value = 1<<30;
	}
}

__global__ void computeSplitBufLoc_kernel(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufLength) return;
	
	const uint dstLoc = bodyPairHash[ind].value;
	if(srcPairs[dstLoc].x == bodyPairHash[ind].key) {
	    splits[dstLoc].x = ind;
	}
	else {
	    splits[dstLoc].y = ind;
	}
}

__global__ void countBody_kernel(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dstCount[ind] = 0;
	
	const uint a = srcInd[ind].key;
	
	int isFirst = 0;
	
	if(ind < 1) isFirst = 1;
	else if(srcInd[ind - 1].key != a) isFirst = 1;
	
	if(!isFirst) return;
	
	dstCount[ind] = 1;

	unsigned cur = ind;
// check backward
	for(;;) {
	    if(cur == maxInd - 1) return;
	    cur++;
	    if(srcInd[cur].key != a) return;
	    dstCount[ind]++;
	}	
}

__global__ void computeSplitInvMass_kernel(float * invMass, 
                                        uint * bodyCount, 
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	uint n = getBodyCountAt(ind, bodyCount);
	
	invMass[ind] = 1.f * (float)n;
}

__global__ void setContactConstraint_kernel(float3 * relLinVel,
                                        float3 * angVelA,
                                        float3 * angVelB,
                                        float * lambda,
                                        float * invMass, 
                                        float * splitInvMass, 
                                        uint2 * splits, 
                                        uint2 * pairs,
                                        float3 * srcPos,
                                        float3 * srcVel,
                                        uint4 * indices,
                                        uint * pointStarts,
                                        uint * indexStarts,
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint2 massInd = splits[ind];
	
	float invMassA = splitInvMass[massInd.x];
	float invMassB = splitInvMass[massInd.y];
	
	invMass[ind] = 1.f / (invMassA + invMassB);
	lambda[ind] = 0.f;
	
	float3 linearVelocityA, linearVelocityB;
	computeBodyLinearVelocity(pointStarts, indexStarts, indices, pairs[ind], srcVel, 
	    linearVelocityA, linearVelocityB);
		
	relLinVel[ind] = float3_difference(linearVelocityB, linearVelocityA);
// omega = r cross v
// v = omega cross r
	angVelA[ind] = make_float3(0.f, 0.f, 0.f);
	angVelB[ind] = make_float3(0.f, 0.f, 0.f);
}

__global__ void clearDeltaVelocity_kernel(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	deltaLinVel[ind] = make_float3(0.f, 0.f, 0.f);
	deltaAngVel[ind] = make_float3(0.f, 0.f, 0.f);
}

__global__ void stopAtContact_kernel(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * pointStarts,
                        uint * indexStarts,
                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].x);
	const uint4 ib = computePointIndex(pointStarts, indexStarts, indices, pairs[ind].y);
	
	dstVelocity[ia.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ia.w] = make_float3(0.f, 0.f, 0.f);
	
	dstVelocity[ib.x] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.y] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.z] = make_float3(0.f, 0.f, 0.f);
	dstVelocity[ib.w] = make_float3(0.f, 0.f, 0.f);
}

__global__ void solveContact_kernel(float3 * deltaLinVel,
                        float3 * deltaAngVel,
	                    uint2 * splits,
	                    float * splitMass,
	                    float3 * srcVelocity,
                    uint2 * pairs,
                    uint4 * indices,
                    uint * pointStarts,
                    uint * indexStarts,
                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	float3 linearVelocityA, linearVelocityB;
	computeBodyLinearVelocity(pointStarts, indexStarts, indices, pairs[ind], srcVelocity, 
	    linearVelocityA, linearVelocityB);
	
	const uint2 dstInd = splits[ind];
	float massA = splitMass[dstInd.x];
	float massB = splitMass[dstInd.y];
	
	collide(linearVelocityA, linearVelocityB,
	        massA, massB,
	        deltaLinVel[dstInd.x],
	        deltaLinVel[dstInd.y]
	        );
}

extern "C" {
    
void simpleContactSolverWriteContactIndex(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufferLength, 512);
    dim3 grid(nblk, 1, 1);
    
    writeContactIndex_kernel<<< grid, block >>>(dstInd, 
                                                srcInd,
                                                n, bufferLength);
}

void simpleContactSolverComputeSplitBufLoc(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitBufLoc_kernel<<< grid, block >>>(splits, 
                                        srcPairs, 
                                        bodyPairHash, 
                                        bufLength);
}

void simpleContactSolverCountBody(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint num)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(num, 512);
    dim3 grid(nblk, 1, 1);
    
    countBody_kernel<<< grid, block >>>(dstCount,
                                     srcInd, 
                                       num);
}

void simpleContactSolverComputeSplitInverseMass(float * invMass, 
                                        uint * bodyCount, 
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitInvMass_kernel<<< grid, block >>>(invMass,
                                     bodyCount, 
                                       bufLength);
}

void simpleContactSolverSetContactConstraint(float3 * relLinVel,
                                        float3 * angVelA,
                                        float3 * angVelB,
                                        float * lambda,
                                        float * invMass, 
                                        float * splitInvMass, 
                                        uint2 * splits, 
                                        uint2 * pairs,
                                        float3 * pos,
                                        float3 * vel,
                                        uint4 * ind,
                                        uint * perObjPointStart,
                                        uint * perObjectIndexStart,
                                        uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    setContactConstraint_kernel<<< grid, block >>>(relLinVel,
                                        angVelA,
                                        angVelB,
                                        lambda,
                                        invMass, 
                                        splitInvMass, 
                                        splits, 
                                        pairs,
                                        pos,
                                        vel,
                                        ind,
                                        perObjPointStart,
                                        perObjectIndexStart,
                                        numContacts);
}

void simpleContactSolverClearDeltaVelocity(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    clearDeltaVelocity_kernel<<< grid, block >>>(deltaLinVel,
                                     deltaAngVel, 
                                       bufLength);
}

void simpleContactSolverStopAtContact(float3 * dstVelocity,
                        uint2 * pairs,
                        uint4 * indices,
                        uint * objectPointStarts,
                        uint * objectIndexStarts,
                        uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    stopAtContact_kernel<<< grid, block >>>(dstVelocity, 
                        pairs,
                        indices,
                        objectPointStarts,
                        objectIndexStarts,
                        numContacts);
}

void simpleContactSolverSolveContact(float3 * deltaLinVel,
	                    float3 * deltaAngVel,
	                    uint2 * splits,
	                    float * splitMass,
	                    float3 * srcVelocity,
                    uint2 * pairs,
                    uint4 * indices,
                    uint * objectPointStarts,
                    uint * objectIndexStarts,
                    uint numContacts)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numContacts, 512);
    dim3 grid(nblk, 1, 1);
    
    solveContact_kernel<<< grid, block >>>(deltaLinVel,
	                    deltaAngVel,
	                    splits,
	                    splitMass,
	                    srcVelocity,
                        pairs,
                        indices,
                        objectPointStarts,
                        objectIndexStarts,
                        numContacts);
}

}
