#include "hip/hip_runtime.h"
#include "narrowphase_implement.h"
#include <bvh_math.cu>
#include <gjk_math.cu>
#include <CudaBase.h>

#define GJK_BLOCK_SIZE 64

inline __device__ void extractTetrahedron(MovingTetrahedron & tet, uint start, const uint4 & vertices, float3 * pos, float3 * vel)
{
    uint ind = start + vertices.x;
    tet.p[0] = pos[ind];
    tet.v[0] = vel[ind];
    ind = start + vertices.y;
    tet.p[1] = pos[ind];
    tet.v[1] = vel[ind];
    ind = start + vertices.z;
    tet.p[2] = pos[ind];
    tet.v[2] = vel[ind];
    ind = start + vertices.w;
    tet.p[3] = pos[ind];
    tet.v[3] = vel[ind];
}

inline __device__ void progressTetrahedron(TetrahedronProxy & prx, const MovingTetrahedron & tet, float h)
{
    prx.p[0] = float3_add(tet.p[0], scale_float3_by(tet.v[0], h));
    prx.p[1] = float3_add(tet.p[1], scale_float3_by(tet.v[1], h));
    prx.p[2] = float3_add(tet.p[2], scale_float3_by(tet.v[2], h));
    prx.p[3] = float3_add(tet.p[3], scale_float3_by(tet.v[3], h));
}

__global__ void computeSeparateAxis_kernel(ContactData * dstContact,
    uint2 * pairs,
    float3 * pos, float3 * vel, 
    uint4* tetrahedron, 
    uint * pointStart, uint * indexStart,
    uint maxInd)
{
    __shared__ Simplex sS[GJK_BLOCK_SIZE];
    __shared__ TetrahedronProxy sPrxA[GJK_BLOCK_SIZE];
	__shared__ TetrahedronProxy sPrxB[GJK_BLOCK_SIZE];
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;

	if(ind >= maxInd) return;
	
	uint objA = extractObjectInd(pairs[ind].x);
	uint objB = extractObjectInd(pairs[ind].y);
	uint elmA = extractElementInd(pairs[ind].x);
	uint elmB = extractElementInd(pairs[ind].y);
	
	MovingTetrahedron tA;
	MovingTetrahedron tB;
	
	extractTetrahedron(tA, pointStart[objA], tetrahedron[indexStart[objA] + elmA], pos, vel);
	extractTetrahedron(tB, pointStart[objB], tetrahedron[indexStart[objB] + elmB], pos, vel);
	
	progressTetrahedron(sPrxA[threadIdx.x], tA, 0.01667f);
	progressTetrahedron(sPrxB[threadIdx.x], tB, 0.01667f);

	ClosestPointTestContext ctc;
	BarycentricCoordinate coord;
	
	computeSeparateDistance(sS[threadIdx.x], sPrxA[threadIdx.x], sPrxB[threadIdx.x], GJK_THIN_MARGIN, ctc, dstContact[ind].separateAxis, 
	    coord);
	
	interpolatePointAB(sS[threadIdx.x], coord, dstContact[ind].localA, dstContact[ind].localB);
}

__global__ void computeTimeOfImpact_kernel(ContactData * dstContact,
    uint2 * pairs,
    float3 * pos, float3 * vel, 
    uint4* tetrahedron, 
    uint * pointStart, uint * indexStart,
    uint maxInd)
{
    __shared__ Simplex sS[GJK_BLOCK_SIZE];
    __shared__ TetrahedronProxy sPrxA[GJK_BLOCK_SIZE];
	__shared__ TetrahedronProxy sPrxB[GJK_BLOCK_SIZE];
	unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;

	if(ind >= maxInd) return;
	
	uint objA = extractObjectInd(pairs[ind].x);
	uint objB = extractObjectInd(pairs[ind].y);
	uint elmA = extractElementInd(pairs[ind].x);
	uint elmB = extractElementInd(pairs[ind].y);
	
	MovingTetrahedron tA;
	MovingTetrahedron tB;
	
	extractTetrahedron(tA, pointStart[objA], tetrahedron[indexStart[objA] + elmA], pos, vel);
	extractTetrahedron(tB, pointStart[objB], tetrahedron[indexStart[objB] + elmB], pos, vel);
	
	progressTetrahedron(sPrxA[threadIdx.x], tA, 0.01667f);
	progressTetrahedron(sPrxB[threadIdx.x], tB, 0.01667f);

	ClosestPointTestContext ctc;
	BarycentricCoordinate coord;

	computeSeparateDistance(sS[threadIdx.x], sPrxA[threadIdx.x], sPrxB[threadIdx.x], 0.f, ctc, dstContact[ind].separateAxis, 
	    coord);
	
	if(dstContact[ind].separateAxis.w < 1.f) {
	    return;
	}
	
	int i = 0;
	while (i<GJK_MAX_NUM_ITERATIONS) {
	    computeSeparateDistance(sS[threadIdx.x], sPrxA[threadIdx.x], sPrxB[threadIdx.x], GJK_THIN_MARGIN, ctc, dstContact[ind].separateAxis, 
	        coord); 
	    
	    if(dstContact[ind].separateAxis.w < 1.f) {
	        break;
	    }
	    
	    i++;   
	}
	interpolatePointAB(sS[threadIdx.x], coord, dstContact[ind].localA, dstContact[ind].localB);
}

extern "C" {

void narrowphaseComputeSeparateAxis(ContactData * dstContact,
		uint2 * pairs,
		float3 * pos,
		float3 * vel,
		uint4 * ind,
		uint * pointStart, uint * indexStart,
		uint numOverlappingPairs)
{
    dim3 block(GJK_BLOCK_SIZE, 1, 1);
    unsigned nblk = iDivUp(numOverlappingPairs, GJK_BLOCK_SIZE);
    dim3 grid(nblk, 1, 1);
    
    computeSeparateAxis_kernel<<< grid, block >>>(dstContact, pairs, pos, vel, ind, pointStart, indexStart, numOverlappingPairs);
}

void narrowphaseComputeTimeOfImpact(ContactData * dstContact,
		uint2 * pairs,
		float3 * pos,
		float3 * vel,
		uint4 * ind,
		uint * pointStart, uint * indexStart, 
		uint numOverlappingPairs)
{   
    dim3 block(GJK_BLOCK_SIZE, 1, 1);
    unsigned nblk = iDivUp(numOverlappingPairs, GJK_BLOCK_SIZE);
    dim3 grid(nblk, 1, 1);
    
    computeTimeOfImpact_kernel<<< grid, block >>>(dstContact, pairs, pos, vel, ind, pointStart, indexStart, numOverlappingPairs);
}

}
