#include "hip/hip_runtime.h"
#ifndef _MATRIX_MATH_H_
#define _MATRIX_MATH_H_

#include "bvh_common.h"

inline __device__ void fill_mat44(mat44 & m, float3 a, float3 b, float3 c, float3 d)
{
    m.v[0] = make_float4(a.x, a.y, a.z, 1.0f);
    m.v[1] = make_float4(b.x, b.y, b.z, 1.0f);
    m.v[2] = make_float4(c.x, c.y, c.z, 1.0f);
    m.v[3] = make_float4(d.x, d.y, d.z, 1.0f);
}

inline __device__ float determinant33( float a, float b, float c, float d, float e, float f, float g, float h, float i )
{
    return ( a*( e*i - h*f ) - b*( d*i - g*f ) + c*( d*h - g*e ) );
}

inline __device__ float determinant44(const mat44 & M)
{
    return  ( M.v[0].x * determinant33( M.v[1].y, M.v[2].y, M.v[3].y, M.v[1].z, M.v[2].z, M.v[3].z, M.v[1].w, M.v[2].w, M.v[3].w )
			- M.v[1].x * determinant33( M.v[0].y, M.v[2].y, M.v[3].y, M.v[0].z, M.v[2].z, M.v[3].z, M.v[0].w, M.v[2].w, M.v[3].w )
			+ M.v[2].x * determinant33( M.v[0].y, M.v[1].y, M.v[3].y, M.v[0].z, M.v[1].z, M.v[3].z, M.v[0].w, M.v[1].w, M.v[3].w )
			- M.v[3].x * determinant33( M.v[0].y, M.v[1].y, M.v[2].y, M.v[0].z, M.v[1].z, M.v[2].z, M.v[0].w, M.v[1].w, M.v[2].w ) );

}

#endif
