#include "hip/hip_runtime.h"
#ifndef _GJK_MATH_H_
#define _GJK_MATH_H_

#include "bvh_common.h"
#include "bvh_math.cu"
#include "barycentric.cu"

#define GJK_MAX_NUM_ITERATIONS 32
struct ClosestPointTestContext {
    float3 referencePoint;
    float3 closestPoint;
    float closestDistance;
};

struct Simplex {
    float3 p[4];
	float3 pA[4];
	float3 pB[4];
	int dimension;
};

struct MovingTetrahedron {
    float3 p[4];
    float3 v[4];
};

struct TetrahedronProxy {
    float3 p[4];
};

inline __device__ void resetSimplex(Simplex & s)
{ s.dimension = 0; }

inline __device__ void addToSimplex(Simplex & s, float3 p, float3 localA, float3 localB)
{
    if(s.dimension < 1) {
        s.p[0] = p;
        s.pA[0] = localA;
        s.pB[0] = localB;
        s.dimension = 1;
    }
    else if(s.dimension < 2) {
		if(distance2_between(p, s.p[0]) > 1e-6) {
		    s.p[1] = p;
		    s.pA[1] = localA;
		    s.pB[1] = localB;
		    s.dimension = 2;
		}
    }
    else if(s.dimension < 3) {
		if(!isTriangleDegenerate(s.p[0], s.p[1], p)) {
		    s.p[2] = p;
		    s.pA[2] = localA;
		    s.pB[2] = localB;
		    s.dimension = 3;
		}
    }
    else {
        if(!isTetrahedronDegenerate(s.p[0], s.p[1], s.p[2], p)) {
		    s.p[3] = p;
		    s.pA[3] = localA;
		    s.pB[3] = localB;
		    s.dimension = 4;
		}
    }
}

inline __device__ float3 initialPoint(const TetrahedronProxy & tet, const float3 & ref)
{
    float3 r = float3_difference(tet.p[0], ref);
    
    if(float3_length2(r) < 1e-6)
        r = float3_difference(tet.p[1], ref);
    
    return r;
}

inline __device__ float3 supportPoint(TetrahedronProxy tet, float3 ref, float margin, float3 & localP)
{
    float maxDotv = -1e8;
    float dotv;
    
    float3 dMargin = scale_float3_by(float3_normalize(ref), margin);
    float3 res, wp;
    
    int i;
    
    float3 center = tet.p[0];
    for(i=1; i<4; i++) {
        center = float3_add(center, tet.p[i]);
    }
    center = scale_float3_by(center,0.25f);
    
    for(i=0; i<4; i++) {
        wp = float3_add(tet.p[i], dMargin);
        dotv = float3_dot(wp, ref);
        if(dotv > maxDotv) {
            maxDotv = dotv;
            res = wp;
            localP = float3_difference(tet.p[i], center);
        }
    }
    
    return res;
}

inline __device__ int isPointInsideSimplex(Simplex & s, float3 p)
{
    if(s.dimension > 3) {
        return pointInsideTetrahedronTest(p, s.p);
    }
    return 0;
}

inline __device__ void computeClosestPointOnLine(float3 p, float3 * v, ClosestPointTestContext & result)
{
    float3 vr = float3_difference(p, v[0]);
    float3 v1 = float3_difference(v[1], v[0]);
	float dr = float3_length(vr);
	if(dr < 1e-6) {
        result.closestPoint = v[0];
		result.closestDistance = 0.f;
        return;
    }
	
	float d1 = float3_length(v1);
	vr = float3_normalize(vr);
	v1 = float3_normalize(v1);
	float vrdv1 = float3_dot(vr, v1) * dr;
	if(vrdv1 < 0.f) vrdv1 = 0.f;
	if(vrdv1 > d1) vrdv1 = d1;
	
	v1 = float3_add(v[0], scale_float3_by(v1, vrdv1));
	float dc = distance_between(v1, p);
	
	if(dc < result.closestDistance) {
	    result.closestPoint = v1;
	    result.closestDistance = dc;
	}
}

// http://mathworld.wolfram.com/Point-PlaneDistance.html

inline __device__ float3 projectPointOnPlane(float3 p, float3 v, float3 nor)
{
    float t = float3_dot(nor, v) - float3_dot(nor, p);
    return float3_add(p, scale_float3_by(nor, t));
}

inline __device__ void computeClosestPointOnTriangle(float3 p, float3 * v, ClosestPointTestContext & result)
{
    float3 nor = triangleNormal(v);
    float3 onplane = projectPointOnPlane(p, v[0], nor);
    
    if(pointInsideTriangleTest(onplane, nor, v)) {
        float d = distance_between(p, onplane);
        if(d < result.closestDistance) {
            result.closestPoint = onplane;
            result.closestDistance = d;
        }
        return;
    }
    
    computeClosestPointOnLine(p, v, result);
    float3 line[2];
    line[0] = v[1];
    line[1] = v[2];
    computeClosestPointOnLine(p, line, result);
    line[0] = v[2];
    line[1] = v[0];
    computeClosestPointOnLine(p, line, result);
}

inline __device__ void computeClosestPointOnTetrahedron(float3 p, float3 * v, ClosestPointTestContext & result)
{
	computeClosestPointOnTriangle(p, v, result);
	
	float3 pr[3];
	pr[0] = v[0];
	pr[1] = v[1];
	pr[2] = v[3];
	computeClosestPointOnTriangle(p, pr, result);
	
	pr[0] = v[0];
	pr[1] = v[2];
	pr[2] = v[3];
	computeClosestPointOnTriangle(p, pr, result);
	
	pr[0] = v[1];
	pr[1] = v[2];
	pr[2] = v[3];
	computeClosestPointOnTriangle(p, pr, result);
}

inline __device__ void computeClosestPointOnSimplex(Simplex & s, float3 p, ClosestPointTestContext & ctc)
{
    ctc.closestDistance = 1e10;

    if(s.dimension < 2) {
        ctc.closestPoint = s.p[0];
        ctc.closestDistance = distance_between(p, s.p[0]);
    }
    else if(s.dimension < 3) {
        computeClosestPointOnLine(p, s.p, ctc);
    }
    else if(s.dimension < 4) {
        computeClosestPointOnTriangle(p, s.p, ctc);
    }
    else {
        computeClosestPointOnTetrahedron(p, s.p, ctc);
    }
}

inline __device__ void computeClosestPointOnSimplex(Simplex & s, ClosestPointTestContext & ctc)
{
    computeClosestPointOnSimplex(s, ctc.referencePoint, ctc);
}

inline __device__ void computeContributionSimplex(BarycentricCoordinate & dst, const Simplex & s, const float3 & q)
{
    if(s.dimension < 2) {
        dst = make_float4(1.f, -1.f, -1.f, -1.f);
    }
    else if(s.dimension < 3) {
        dst = getBarycentricCoordinate2(q, s.p);
    }
    else if(s.dimension < 4) {
        dst = getBarycentricCoordinate3(q, s.p);
    }
    else {
        dst = getBarycentricCoordinate4(q, s.p);
    }
}

inline __device__ void interpolatePointAB(Simplex & s,
                                            const BarycentricCoordinate & contributes, 
                                            float3 & pA, float3 & pB)
{
	pA = make_float3(0.f, 0.f, 0.f);
	pB = make_float3(0.f, 0.f, 0.f);
	const float * wei = &contributes.x;
	int i;
	for(i =0; i < s.dimension; i++) {
		if(wei[i] > 1e-5) {
		    pA = float3_add(pA, scale_float3_by(s.pA[i], wei[i]));
			pB = float3_add(pB, scale_float3_by(s.pB[i], wei[i]));
		}
	}
}

inline __device__ void compareAndSwap(float * key, float3 * v1, float3* v2, float3 * v3, int a, int b)
{
    if(key[a] < key[b]) {
        float ck = key[a];
        key[a] = key[b];
        key[b] = ck;
        
        float3 cv = v1[a];
        v1[a] = v1[b];
        v1[b] = cv;
        
        cv = v2[a];
        v2[a] = v2[b];
        v2[b] = cv;
        
        cv = v3[a];
        v3[a] = v3[b];
        v3[b] = cv;
    }
}

inline __device__ void smallestSimplex(Simplex & s, BarycentricCoordinate & contributes)
{
	if(s.dimension < 2) return;
	
	float * bar = &contributes.x;
	
	compareAndSwap(bar, s.p, s.pA, s.pB, 0, 2);
	compareAndSwap(bar, s.p, s.pA, s.pB, 1, 3);
	compareAndSwap(bar, s.p, s.pA, s.pB, 0, 1);
	compareAndSwap(bar, s.p, s.pA, s.pB, 2, 3);
	compareAndSwap(bar, s.p, s.pA, s.pB, 1, 2);

	s.dimension = 0;
	
	if(bar[0] > 1e-5) s.dimension++;
	if(bar[1] > 1e-5) s.dimension++;
	if(bar[2] > 1e-5) s.dimension++;
	if(bar[3] > 1e-5) s.dimension++;
}

inline __device__ void computeSeparateDistance(Simplex & s, 
                                               const TetrahedronProxy & prxA,
                                               const TetrahedronProxy & prxB,
                                               ClosestPointTestContext & ctc,
                                               float4 & separateAxis,
                                               BarycentricCoordinate & coord)
{
    resetSimplex(s);

	float3 v = initialPoint(prxA, ctc.referencePoint);
	
	float3 w, supportA, supportB, localA, localB;
	float margin = 0.02f;
	float v2;
	int i = 0;
	
	while(i<GJK_MAX_NUM_ITERATIONS) {
	    supportA = supportPoint(prxA, float3_reverse(v), margin, localA);
	    supportB = supportPoint(prxB, v, margin, localB);
	    
	    w = float3_difference(supportA, supportB);
	    
	    v2 = float3_length2(v);
	    if((v2 - float3_dot(w, v)) < 0.0001f * v2) {
	        return;
	    }
	    
	    addToSimplex(s, w, localA, localB);
	    
	    if(isPointInsideSimplex(s, ctc.referencePoint)) {
	        separateAxis.w = 0.f;
	        return;
	    }
	    
	    computeClosestPointOnSimplex(s, ctc);
	    
	    v = float3_difference(ctc.closestPoint, ctc.referencePoint);
	    separateAxis = make_float4(v.x, v.y, v.z, 1.f);
	    
	    computeContributionSimplex(coord, s, ctc.closestPoint);
	    
	    smallestSimplex(s, coord);
	    
	    i++;
	}
}

inline __device__ void checkClosestDistance(Simplex & s, 
                                        const TetrahedronProxy & prxA,
                                        const TetrahedronProxy & prxB,
                                        ClosestPointTestContext & result,
                                        float4 & dstSA,
                                        float3 & dstPA,
                                        float3 & dstPB,
                                        BarycentricCoordinate & coord)
{
    float3 cenA = prxA.p[0];
	cenA = float3_add(cenA, prxA.p[1]);
	cenA = float3_add(cenA, prxA.p[2]);
	cenA = float3_add(cenA, prxA.p[3]);
	cenA = scale_float3_by(cenA, 0.25f);
	
	float3 cenB = prxB.p[0];
	cenB = float3_add(cenB, prxB.p[1]);
	cenB = float3_add(cenB, prxB.p[2]);
	cenB = float3_add(cenB, prxB.p[3]);
	cenB = scale_float3_by(cenB, 0.25f);
	
    resetSimplex(s);
	
	float3 la, lb;
	la = float3_difference(prxA.p[0], cenA);
	lb = float3_difference(prxB.p[0], cenB);
	addToSimplex(s, prxA.p[0], la, lb);
	la = float3_difference(prxA.p[1], cenA);
	lb = float3_difference(prxB.p[1], cenB);
	addToSimplex(s, prxA.p[1], la, lb);
	la = float3_difference(prxA.p[2], cenA);
	lb = float3_difference(prxB.p[2], cenB);
	// addToSimplex(s, prxA.p[2], la, lb);
	la = float3_difference(prxA.p[3], cenA);
	lb = float3_difference(prxB.p[3], cenB);
	addToSimplex(s, prxA.p[3], la, lb);

	computeClosestPointOnSimplex(s, cenB, result);
	
	float3 d = float3_difference(result.closestPoint, cenB);
	dstSA = make_float4(d.x, d.y, d.z, 1.f);
	
	computeContributionSimplex(coord, s, result.closestPoint);
	
	interpolatePointAB(s, coord, dstPA, dstPB);
}

#endif        //  #ifndef _GJK_MATH_H_

