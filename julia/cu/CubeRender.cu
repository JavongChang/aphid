#include "CubeRender.cuh"

namespace cuber {

void setRenderRect(int * src)
{ hipMemcpyToSymbol(HIP_SYMBOL(c_renderRect), src, 16); }

void setFrustum(float * src)
{ hipMemcpyToSymbol(HIP_SYMBOL(c_frustumVec), src, 72); }

void render(uint * pix,
            float * depth,
            int blockx,
            int gridx, int gridy)
{
    dim3 block(blockx, blockx, 1);
    dim3 grid(gridx, gridy, 1);
    
    oneCube_kernel<<< grid, block >>>(pix, 
        depth);
}

const float cubefaces[] = {
-1, 0, 0,
 1, 0, 0,
 0,-1, 0,
 0, 1, 0,
 0, 0,-1,
 0, 0, 1
};

void setBoxFaces()
{ hipMemcpyToSymbol(HIP_SYMBOL(c_ray_box_face), cubefaces, 72); }

}
