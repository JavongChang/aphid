#include "hip/hip_runtime.h"
#include "cuFemTetrahedron_implement.h"
#include "cuFemMath.cu"

__global__ void internalForce_kernel(float3 * dst,
    float d16, float d17, float d18,
                                    float3 * pos,
                                    uint4 * tetvert,
                                    mat33 * orientation,
                                    KeyValuePair * tetraInd,
                                    uint * bufferIndices,
                                    uint maxBufferInd,
                                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	float volume;
	float3 B[4];
	float3 pj, force, sum;
	mat33 Ke, Re;
	uint iTet, i, j;
	uint cur = bufferIndices[ind];
	uint lastTet = 94967295;
	for(;;) {
	    if(tetraInd[cur].key != ind) break;
	    
	    extractTetij(tetraInd[cur].value, iTet, i, j);
	    
	    if(lastTet != iTet) {
	        if(lastTet != 94967295) {
	            mat33_float3_prod(force, Re, sum);
	            float3_minus_inplace(dst[ind], force);
	        }
	        Re = orientation[iTet];
	        calculateBandVolume(B, volume, pos, tetvert[iTet]);
	        float3_set_zero(sum);
	        lastTet = iTet;
	    }

	    calculateKe(Ke, B, d16, d17, d18, volume, i, j);	    
	    
	    uint * tetv = &tetvert[iTet].x;
	    pj = pos[tetv[j]];
		
	    mat33_float3_prod(force, Ke, pj);	    
	    float3_add_inplace(sum, force);

	    cur++;
	    if(cur >= maxBufferInd) break;
	}
	
	mat33_float3_prod(force, Re, sum);
	float3_minus_inplace(dst[ind], force);
}

__global__ void resetForce_kernel(float3 * dst,
    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	float3_set_zero(dst[ind]);
}

__global__ void resetStiffnessMatrix_kernel(mat33* dst, 
                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	set_mat33_zero(dst[ind]);
}

__global__ void stiffnessAssembly_kernel(mat33 * dst,
                                        float d16, float d17, float d18,
                                        float3 * pos,
                                        uint4 * tetv,
                                        mat33 * orientation,
                                        KeyValuePair * tetraInd,
                                        uint * bufferIndices,
                                        uint maxBufferInd,
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	float volume;
	float3 B[4];
	mat33 Ke, Re, ReT, tmp, tmpT;
	uint iTet, i, j;
	uint cur = bufferIndices[ind];
	for(;;) {
	    if(tetraInd[cur].key != ind) break;
	    
	    extractTetij(tetraInd[cur].value, iTet, i, j);
	    
	    calculateBandVolume(B, volume, pos, tetv[iTet]);
		
	    calculateKe(Ke, B, d16, d17, d18, volume, i, j);

	    Re = orientation[iTet];
	    mat33_transpose(ReT, Re);
	    
	    mat33_cpy(tmp, Re);
	    mat33_mult(tmp, Ke);
	    mat33_mult(tmp, ReT);
	        
	    mat33_add(dst[ind], tmp);
	    
	    if(j>i) {
	        mat33_transpose(tmpT, tmp);
	        mat33_add(dst[ind], tmpT);
	    }
	    
	    cur++;
	    if(cur >= maxBufferInd) break;
	}
}

__global__ void resetRe_kernel(mat33* dst, 
                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	set_mat33_identity(dst[ind]);
}

__global__ void calculateRe_kernel(mat33 * dst, 
                                    float3 * pos, 
                                    float3 * pos0,
                                    uint4 * indices,
                                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	uint4 & t = indices[ind];
	
	float3 pnt[4];
	tetrahedronP(pnt, pos0, t);
	float3 e01, e02, e03;
	tetrahedronEdge(e01, e02, e03, pnt); 
	
	float div6V = 1.f / tetrahedronVolume(e01, e02, e03) * 6.f;

	tetrahedronP(pnt, pos, t);
	float3 e1, e2, e3;
	tetrahedronEdge(e1, e2, e3, pnt); 
	float3 n1 = scale_float3_by(float3_cross(e2, e3), div6V);
	float3 n2 = scale_float3_by(float3_cross(e3, e1), div6V);
	float3 n3 = scale_float3_by(float3_cross(e1, e3), div6V);
	
	mat33 & Re = dst[ind];
	Re.v[0].x = e01.x * n1.x + e02.x * n2.x + e03.x * n3.x;  
	Re.v[1].x = e01.x * n1.y + e02.x * n2.y + e03.x * n3.y;   
	Re.v[2].x = e01.x * n1.z + e02.x * n2.z + e03.x * n3.z;

    Re.v[0].y = e01.y * n1.x + e02.y * n2.x + e03.y * n3.x;  
	Re.v[1].y = e01.y * n1.y + e02.y * n2.y + e03.y * n3.y;   
	Re.v[2].y = e01.y * n1.z + e02.y * n2.z + e03.y * n3.z;

    Re.v[0].z = e01.z * n1.x + e02.z * n2.x + e03.z * n3.x;  
	Re.v[1].z = e01.z * n1.y + e02.z * n2.y + e03.z * n3.y;  
	Re.v[2].z = e01.z * n1.z + e02.z * n2.z + e03.z * n3.z;
	
	mat33_orthoNormalize(Re);
}

extern "C" {
void cuFemTetrahedron_resetRe(mat33 * d, uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    resetRe_kernel<<< grid, block >>>(d, maxInd);
}

void cuFemTetrahedron_calculateRe(mat33 * dst, 
                                    float3 * pos, 
                                    float3 * pos0,
                                    uint4 * indices,
                                    uint maxInd)
{
    int tpb = CudaBase::LimitNThreadPerBlock(24, 50);
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(maxInd, tpb);
    dim3 grid(nblk, 1, 1);
    
    calculateRe_kernel<<< grid, block >>>(dst, 
                                       pos, 
                                       pos0,
                                       indices,
                                       maxInd);
}

void cuFemTetrahedron_resetStiffnessMatrix(mat33 * dst,
                                    uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    resetStiffnessMatrix_kernel<<< grid, block >>>(dst, 
                                        maxInd);
}

void cuFemTetrahedron_stiffnessAssembly(mat33 * dst,
                                        float3 * pos,
                                        uint4 * vert,
                                        mat33 * orientation,
                                        KeyValuePair * tetraInd,
                                        uint * bufferIndices,
                                        uint maxBufferInd,
                                        uint maxInd)
{
    int tpb = CudaBase::LimitNThreadPerBlock(24, 50);
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(maxInd, tpb);
    dim3 grid(nblk, 1, 1);
    
    stiffnessAssembly_kernel<<< grid, block >>>(dst,
        1.f, 2.f, 3.f,
                                            pos,
                                            vert,
                                            orientation,
                                            tetraInd,
                                            bufferIndices,
                                            maxBufferInd,
                                            maxInd);
}

void cuFemTetrahedron_resetForce(float3 * dst,
                                    uint maxInd)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(maxInd, 512);
    dim3 grid(nblk, 1, 1);
    
    resetForce_kernel<<< grid, block >>>(dst, maxInd);
}

void cuFemTetrahedron_internalForce(float3 * dst,
                                    float3 * pos,
                                    uint4 * tetvert,
                                    mat33 * orientation,
                                    KeyValuePair * tetraInd,
                                    uint * bufferIndices,
                                    uint maxBufferInd,
                                    uint maxInd)
{
    int tpb = CudaBase::LimitNThreadPerBlock(34, 50);
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(maxInd, tpb);
    dim3 grid(nblk, 1, 1);
    
    internalForce_kernel<<< grid, block >>>(dst,
        1.f, 2.f, 3.f,
                                            pos,
                                            tetvert,
                                            orientation,
                                            tetraInd,
                                            bufferIndices,
                                            maxBufferInd,
                                            maxInd);
}

}
