#include "hip/hip_runtime.h"
#ifndef _RADIXSORT_KERNEL_H_
#define _RADIXSORT_KERNEL_H_

#include <hip/hip_runtime.h>


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

#include <cutil_inline.h>

#include <shrUtils.h>
#include <algorithm>
#include <time.h>
#include <limits.h>

#include "solver_implement.h"
#define TILE_I 8
#define TILE_J 8

texture<float, hipTextureType2D> scalarTex;

__global__ void
show_scalar(float *id, unsigned char *od, int w, int h)
{
    uint x = blockIdx.x*TILE_I + threadIdx.x;
    uint y = blockIdx.y*TILE_J + threadIdx.y;
    uint i = y * w + x;

    float luminance = id[i] * 255;
    if(luminance > 255.f)
        luminance = 255.f;

    od[i*3] = od[i*3+1] = od[i*3+2] = (unsigned char)luminance;
}

__global__ void
advect_scalar(float *result, float *u, int w, int h)
{
    uint x = blockIdx.x*TILE_I + threadIdx.x;
    uint y = blockIdx.y*TILE_J + threadIdx.y;
    uint i = y * w + x;

    float tx = (float)x + 0.5 - u[i * 2];
    float ty = (float)y + 0.5 - u[i * 2 + 1];
    result[i] = tex2D(scalarTex, tx, ty);	
    if(result[i] < 0.f) result[i] = 0.f;
    if(result[i] > 1.5f) result[i] = 1.5f;
}

extern "C" 
void showScalarField(int width, int height, float *pScalar, unsigned char *outImage)
{
    const int size = width * height * sizeof(unsigned char) * 3;
	
    unsigned char *d_Out;
    cutilSafeCall( hipMalloc((void **)&d_Out, size) );
    
    const int scalarSize = width * height * sizeof(float);
    float *d_In;
    cutilSafeCall( hipMalloc((void **)&d_In, scalarSize) );
    cutilSafeCall( hipMemcpy(d_In, pScalar, scalarSize, hipMemcpyHostToDevice) );

    dim3 grid = dim3(width/TILE_I, height/TILE_J);
    dim3 block = dim3(TILE_I, TILE_J);
    
    show_scalar<<< grid, block>>>(d_In, d_Out, width, height);
	
    cutilSafeCall( hipMemcpy( outImage, d_Out,  size, hipMemcpyDeviceToHost));
    
    cutilSafeCall( hipFree(d_In));
    cutilSafeCall( hipFree(d_Out)); 

}

extern "C" 
void advectScalarField(int width, int height, float*u, float*field)
{
    const int fieldSize = width * height * sizeof(float);
    
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_array;
    cutilSafeCall( hipMallocArray( &cu_array, &channelDesc, width, height )); 
    cutilSafeCall( hipMemcpyToArray( cu_array, 0, 0, field, fieldSize, hipMemcpyHostToDevice));

    scalarTex.addressMode[0] = hipAddressModeClamp;
    scalarTex.addressMode[1] = hipAddressModeClamp;
    scalarTex.filterMode = hipFilterModeLinear;
    scalarTex.normalized = false;

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray(scalarTex, cu_array, channelDesc));

    float *d_field;
    cutilSafeCall( hipMalloc((void **)&d_field, fieldSize) );
    
    const int uSize = width * height * sizeof(float) * 2;
    float *d_u;
    cutilSafeCall( hipMalloc((void **)&d_u, uSize) );
    cutilSafeCall( hipMemcpy(d_u, u, uSize, hipMemcpyHostToDevice) );
    
    dim3 grid = dim3(width/TILE_I, height/TILE_J);
    dim3 block = dim3(TILE_I, TILE_J);
    advect_scalar<<<grid, block>>>(d_field, d_u, width, height);
    
    cutilSafeCall( hipMemcpy( field, d_field, fieldSize, hipMemcpyDeviceToHost));
    
    cutilSafeCall(hipUnbindTexture(scalarTex));
    cutilSafeCall(hipFreeArray(cu_array));
    cutilSafeCall( hipFree(d_field));
    cutilSafeCall( hipFree(d_u));
}

#endif 