#include "hip/hip_runtime.h"
#include "SahInterface.h"

#include "SahSplit.cuh"

namespace sahsplit {
int doSplitWorks(void * q, int * qelements,
                    int2 * nodes,
                    Aabb * nodeAabbs,
                    int * nodeParents,
                    int * nodeLevels,
                    KeyValuePair * primitiveIndirections,
                    Aabb * primitiveAabbs,
                    KeyValuePair * intermediateIndirections,
                    uint numPrimitives)
{
    simpleQueue::SimpleQueue * queue = (simpleQueue::SimpleQueue *)q;
    simpleQueue::init_kernel<<< 1,32 >>>(queue, qelements);
    
    DataInterface data;
    data.nodes = nodes;
    data.nodeAabbs = nodeAabbs;
    data.nodeParents = nodeParents;
    data.nodeLevels = nodeLevels;
    data.primitiveIndirections = primitiveIndirections;
    data.primitiveAabbs = primitiveAabbs;
    data.intermediateIndirections = intermediateIndirections;
    
    SplitTask task;
    
    const int tpb = 256;
    dim3 block(tpb, 1, 1);
    const unsigned nblk = 1024;
    dim3 grid(nblk, 1, 1);
    
    int lpb = 1 + numPrimitives>>10;
    
    work_kernel<simpleQueue::SimpleQueue, SplitTask, DataInterface, 24, 8, 256><<<grid, block, 16320>>>(queue,
                                task,
                                data,
                                lpb,
                                numPrimitives-1);
                                
    simpleQueue::SimpleQueue result;
    hipError_t err = hipMemcpy(&result, queue, SIZE_OF_SIMPLEQUEUE, hipMemcpyDeviceToHost); 
    if (err != hipSuccess) {
        printf(" cu error %s when retrieving task queue result\n", hipGetErrorString(err));
    }
    
    //printf("q out tail %i\n", result._qouttail);
    return result._qouttail;
}
}
