#include "hip/hip_runtime.h"
#include "sahbvh_implement.h"
#include <bvh_math.cu>
#include "sah_math.cu"

#define ASSIGNE_EMISSIONID_NTHREAD 512
#define ASSIGNE_EMISSIONID_NTHREAD_M1 511
#define ASSIGNE_EMISSIONID_NTHREAD_LOG2 9
#define COMPUTE_BINS_NTHREAD 128

__global__ void computeBins_kernel(SplitBin * splitBins,
                        Aabb * rootAabbs,
                        SplitId * splitIds,
                        Aabb * clusterAabbs,
                        uint numBins,
                        uint numClusters)
{      
    __shared__ int sSide[SAH_MAX_NUM_BINS * COMPUTE_BINS_NTHREAD];
    
    uint ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= numClusters) return;
	
	int * sideVertical = &sSide[SAH_MAX_NUM_BINS * threadIdx.x];
	int * sideHorizontal = &sSide[threadIdx.x];
    
	uint iEmission = splitIds[ind].emissionId;
    
    Aabb rootBox = rootAabbs[iEmission];
    float * boxLow = &rootBox.low.x;
    Aabb clusterBox = clusterAabbs[ind];
    float3 center = centroidOfAabb(clusterBox);
    float * p = &center.x;
    
    const float g = longestSideOfAabb(rootBox) * .003f;
    
    computeSplitSide(sideVertical,
                        0,
                        &rootBox,
                        numBins,
                        p,
                        boxLow);
      
    __syncthreads();
    
    if(threadIdx.x < numBins)
    updateBins(splitBins,
               splitIds,
               clusterAabbs,
               sideHorizontal,
               rootBox.low,
               g,
               0,
               COMPUTE_BINS_NTHREAD,
               numBins,
               numClusters);
    
    computeSplitSide(sideVertical,
                        1,
                        &rootBox,
                        numBins,
                        p,
                        boxLow);
    
    __syncthreads();
    
     if(threadIdx.x < numBins)
     updateBins(splitBins,
               splitIds,
               clusterAabbs,
               sideHorizontal,
               rootBox.low,
               g,
               1,
               COMPUTE_BINS_NTHREAD,
               numBins,
               numClusters);

    computeSplitSide(sideVertical,
                        2,
                        &rootBox,
                        numBins,
                        p,
                        boxLow);
    
    __syncthreads();
    
     if(threadIdx.x < numBins)
     updateBins(splitBins,
               splitIds,
               clusterAabbs,
               sideHorizontal,
               rootBox.low,
               g,
               2,
               COMPUTE_BINS_NTHREAD,
               numBins,
               numClusters);

}

__global__ void resetBins_kernel(SplitBin * splitBins, 
                        EmissionBlock * inEmissions,
                        uint numBins)
{
    if(threadIdx.x >= numBins * 3) return;
    
    uint iEmission = blockIdx.x;
    const uint firstBin = iEmission * numBins * 3;
    
    resetSplitBin(splitBins[firstBin + threadIdx.x]);
}

__global__ void assignEmissionId_kernel(SplitId * splitIds,
        EmissionBlock * inEmissions,
        int2 * rootRanges,
        uint nEmissions)
{
    uint iEmission = blockIdx.x;
    if(iEmission >= nEmissions) return;
    
    uint iRoot = inEmissions[iEmission].root_id;
    const int primitiveRangeBegin = rootRanges[iRoot].x;
    const int primitiveRangeEnd = rootRanges[iRoot].y;
    int numPrimitivesInRange = primitiveRangeEnd - primitiveRangeBegin + 1;
    if(numPrimitivesInRange < 0) return; // invalid range

    int npt = numPrimitivesInRange>>ASSIGNE_EMISSIONID_NTHREAD_LOG2;
    if(numPrimitivesInRange & ASSIGNE_EMISSIONID_NTHREAD_M1) npt++;
    
    int i, j;
    for(i=0; i<npt; i++) {
        j = threadIdx.x * npt + i;
        if(j < numPrimitivesInRange)
            splitIds[primitiveRangeBegin + j].emissionId = iEmission;
    }
}

__global__ void countTreeBits_kernel(uint * nbits, 
                            KeyValuePair * morton,
                            uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
    
    nbits[ind] = 32 - __clz(morton[ind].key);
}

__global__ void writeSortedHash_kernel(KeyValuePair * dst,
							KeyValuePair * src,
							uint * indices,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dst[ind] = src[indices[ind]];
}

__global__ void computeClusterAabbs_kernel(Aabb * clusterAabbs,
            Aabb * primitiveAabbs,
            uint * compressedIndices,
            KeyValuePair * sorted,
            uint * offset,
            uint * runLength,
            uint nRuns)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= nRuns) return;
    
    const uint sortedInd = sorted[ind].value;
	const uint start = offset[ind];
	const uint first = compressedIndices[sortedInd];
	const uint l = runLength[ind];
    
    Aabb box;
    resetAabb(box);
    uint i = 0;
	for(;i<l;i++) 
        expandAabb(box, primitiveAabbs[first + i]);
	
    clusterAabbs[ind] = box;
}

__global__ void decompressIndices_kernel(uint * decompressedIndices,
                    uint * compressedIndices,
					KeyValuePair * sorted,
					uint * offset,
					uint * runLength,
					uint nRuns)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= nRuns) return;
	
	const uint sortedInd = sorted[ind].value;
	const uint start = offset[ind];
	const uint first = compressedIndices[sortedInd];
	const uint l = runLength[ind];
	
	uint i = 0;
	for(;i<l;i++)
	    decompressedIndices[start + i] = first + i;
}

__global__ void copyHash_kernel(KeyValuePair * dst,
					KeyValuePair * src,
					uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dst[ind] = src[ind];
}

__global__ void compressRunHead_kernel(uint * compressed, 
							uint * runHeads,
							uint * indices,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	if(runHeads[ind]) compressed[indices[ind]] = ind;
}

__global__ void computeRunLength_kernel(uint * runLength,
							uint * runHeads,
							KeyValuePair * indices,
							uint nRuns,
							uint nPrimitives,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
    
	if(ind >= nRuns) {
	    runLength[ind] = 0;
	    return;
	}
	
	uint sortedInd = indices[ind].value;
	
	if(sortedInd >= nRuns-1) 
	    runLength[ind] = nPrimitives 
	                    - runHeads[sortedInd];
	else
	    runLength[ind] = runHeads[sortedInd+1] 
	                    - runHeads[sortedInd];
}

__global__ void computeRunHash_kernel(KeyValuePair * compressed, 
						KeyValuePair * morton,
						uint * indices,
                        uint m,
						uint d,
						uint maxElem,
						uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	compressed[ind].value = ind;
	
	if(ind >= maxElem)
	    compressed[ind].key = 1<<(m*3);
	else 
	    compressed[ind].key = (morton[indices[ind]].key) >> d;
}

__global__ void computeRunHead_kernel(uint * blockHeads, 
							KeyValuePair * mortonCode,
							uint d,
							uint maxElem,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	if(ind >= maxElem) {
	    blockHeads[ind] = 0;
	    return;
	}
	
	if(ind < 1) {
	    blockHeads[ind] = 1;
	    return;
	}

	uint clft = mortonCode[ind - 1].key;
	uint crgt = mortonCode[ind].key;
	
	if(clft>>d == crgt>>d) blockHeads[ind] = 0;
	else blockHeads[ind] = 1;
}

extern "C" {
void sahbvh_computeRunHead(uint * blockHeads, 
							KeyValuePair * mortonCode,
							uint d,
							uint n,
							uint bufLength)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(bufLength, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeRunHead_kernel<<< grid, block>>>(blockHeads,
        mortonCode,
        d,
        n,
        bufLength);
}

void sahbvh_computeRunHash(KeyValuePair * compressed, 
						KeyValuePair * morton,
						uint * indices,
                        uint m,
						uint d,
						uint n,
						uint bufLength)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(bufLength, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeRunHash_kernel<<< grid, block>>>(compressed,
        morton,
        indices,
        m,
        d,
        n,
        bufLength);
}

void sahbvh_computeRunLength(uint * runLength,
							uint * runHeads,
							KeyValuePair * indices,
							uint nRuns,
							uint nPrimitives,
							uint bufLength)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(bufLength, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeRunLength_kernel<<< grid, block>>>(runLength,
        runHeads,
        indices,
        nRuns,
        nPrimitives,
        bufLength);
}

void sahbvh_compressRunHead(uint * compressed, 
							uint * runHeads,
							uint * indices,
							uint n)
							
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    compressRunHead_kernel<<< grid, block>>>(compressed,
        runHeads,
        indices,
        n);
}

void sahbvh_copyHash(KeyValuePair * dst,
					KeyValuePair * src,
					uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    copyHash_kernel<<< grid, block>>>(dst,
        src,
        n);
}

void sahbvh_decompressIndices(uint * decompressedIndices,
                    uint * compressedIndices,
					KeyValuePair * sorted,
					uint * offset,
					uint * runLength,
					uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    decompressIndices_kernel<<< grid, block>>>(decompressedIndices,
                                            compressedIndices,
                                            sorted,
                                          offset,
                                          runLength,
                                          n);
}

void sahbvh_writeSortedHash(KeyValuePair * dst,
							KeyValuePair * src,
							uint * indices,
							uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    writeSortedHash_kernel<<< grid, block>>>(dst,
							src,
							indices,
							n);
}

void sahbvh_countTreeBits(uint * nbits, 
                            KeyValuePair * morton,
                            uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    countTreeBits_kernel<<< grid, block>>>(nbits, 
                            morton,
                            n);
}

void sahbvh_computeClusterAabbs(Aabb * clusterAabbs,
            Aabb * primitiveAabbs,
            uint * compressedIndices,
            KeyValuePair * sorted,
            uint * offset,
            uint * runLength,
            uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeClusterAabbs_kernel<<< grid, block>>>(clusterAabbs, 
                            primitiveAabbs,
                            compressedIndices,
                            sorted,
                            offset,
                            runLength,
                            n);
}

void sahbvh_assignEmissionId(SplitId * splitIds,
        EmissionBlock * inEmissions,
        int2 * rootRanges,
        uint numEmissions)
{
    const int tpb = ASSIGNE_EMISSIONID_NTHREAD;
    dim3 block(tpb, 1, 1);
    const int nblk = numEmissions;
    dim3 grid(nblk, 1, 1);
// one block for each emission
// assign emissionIds to each primitive in range
    assignEmissionId_kernel<<< grid, block>>>(splitIds,
        inEmissions,
        rootRanges,
        numEmissions);
}

void sahbvh_resetBins(SplitBin * splitBins, 
                        EmissionBlock * inEmissions,
                        Aabb * rootAabbs,
                        uint numBins,
                        uint numEmissions)
{
    const int tpb = 128;
    dim3 block(tpb, 1, 1);
    const int nblk = numEmissions;
    dim3 grid(nblk, 1, 1);
// one block for each emission
// reset 3 * n bins
    resetBins_kernel<<< grid, block>>>(splitBins, 
        inEmissions,
        numBins);
}

void sahbvh_computeBins(SplitBin * splitBins,
                        Aabb * rootAabbs,
                        SplitId * splitIds,
                        Aabb * clusterAabbs,
                        uint numBins,
                        uint numClusters)
{
    const int tpb = COMPUTE_BINS_NTHREAD;
    dim3 block(tpb, 1, 1);
    const int nblk = iDivUp(numClusters, tpb);
    dim3 grid(nblk, 1, 1);
// one thread for each cluster/primitive
// find bins according to splitId
// atomic update bin contents

    computeBins_kernel<<< grid, block>>>(splitBins,
                        rootAabbs,
                        splitIds,
                        clusterAabbs,
                        numBins,
                        numClusters);
}

void sahbvh_emitSahSplit(EmissionBlock * outEmissions,
	    EmissionBlock * inEmissions,
	    int2 * rootRanges,
	    Aabb * rootAabbs,
	    KeyValuePair * clusterMorton,
        Aabb * clusterAabbs,
        SplitBin * splitBins,
        SplitId * splitIds,
	    uint numClusters,
        uint numBins,
	    uint numEmissions)
{
    sahbvh_assignEmissionId(splitIds,
                            inEmissions,
                            rootRanges,
                            numEmissions);
    
    sahbvh_resetBins(splitBins, 
                        inEmissions,
                        rootAabbs,
                        numBins, 
                        numEmissions);
    
    sahbvh_computeBins(splitBins, 
                        rootAabbs,
                        splitIds,
                        clusterAabbs, 
                        numBins,
                        numClusters);
}

}
