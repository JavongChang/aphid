#include "hip/hip_runtime.h"
#include "sahbvh_implement.h"
#include <bvh_math.cu>

__global__ void countTreeBits_kernel(uint * nbits, 
                            KeyValuePair * morton,
                            uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
    
    nbits[ind] = 32 - __clz(morton[ind].key);
}

__global__ void writeSortedHash_kernel(KeyValuePair * dst,
							KeyValuePair * src,
							uint * indices,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dst[ind] = src[indices[ind]];
}

__global__ void computeClusterAabbs_kernel(Aabb * clusterAabbs,
            Aabb * primitiveAabbs,
            uint * compressedIndices,
            KeyValuePair * sorted,
            uint * offset,
            uint * runLength,
            uint nRuns)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= nRuns) return;
    
    const uint sortedInd = sorted[ind].value;
	const uint start = offset[ind];
	const uint first = compressedIndices[sortedInd];
	const uint l = runLength[ind];
    
    Aabb box;
    resetAabb(box);
    uint i = 0;
	for(;i<l;i++) 
        expandAabb(box, primitiveAabbs[first + i]);
	
    clusterAabbs[ind] = box;
}

__global__ void decompressIndices_kernel(uint * decompressedIndices,
                    uint * compressedIndices,
					KeyValuePair * sorted,
					uint * offset,
					uint * runLength,
					uint nRuns)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= nRuns) return;
	
	const uint sortedInd = sorted[ind].value;
	const uint start = offset[ind];
	const uint first = compressedIndices[sortedInd];
	const uint l = runLength[ind];
	
	uint i = 0;
	for(;i<l;i++)
	    decompressedIndices[start + i] = first + i;
}

__global__ void copyHash_kernel(KeyValuePair * dst,
					KeyValuePair * src,
					uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dst[ind] = src[ind];
}

__global__ void compressRunHead_kernel(uint * compressed, 
							uint * runHeads,
							uint * indices,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	if(runHeads[ind]) compressed[indices[ind]] = ind;
}

__global__ void computeRunLength_kernel(uint * runLength,
							uint * runHeads,
							KeyValuePair * indices,
							uint nRuns,
							uint nPrimitives,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
    
	if(ind >= nRuns) {
	    runLength[ind] = 0;
	    return;
	}
	
	uint sortedInd = indices[ind].value;
	
	if(sortedInd >= nRuns-1) 
	    runLength[ind] = nPrimitives 
	                    - runHeads[sortedInd];
	else
	    runLength[ind] = runHeads[sortedInd+1] 
	                    - runHeads[sortedInd];
}

__global__ void computeRunHash_kernel(KeyValuePair * compressed, 
						KeyValuePair * morton,
						uint * indices,
                        uint m,
						uint d,
						uint maxElem,
						uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	compressed[ind].value = ind;
	
	if(ind >= maxElem)
	    compressed[ind].key = 1<<(m*3);
	else 
	    compressed[ind].key = (morton[indices[ind]].key) >> d;
}

__global__ void computeRunHead_kernel(uint * blockHeads, 
							KeyValuePair * mortonCode,
							uint d,
							uint maxElem,
							uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	if(ind >= maxElem) {
	    blockHeads[ind] = 0;
	    return;
	}
	
	if(ind < 1) {
	    blockHeads[ind] = 1;
	    return;
	}

	uint clft = mortonCode[ind - 1].key;
	uint crgt = mortonCode[ind].key;
	
	if(clft>>d == crgt>>d) blockHeads[ind] = 0;
	else blockHeads[ind] = 1;
}

extern "C" {
void sahbvh_computeRunHead(uint * blockHeads, 
							KeyValuePair * mortonCode,
							uint d,
							uint n,
							uint bufLength)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(bufLength, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeRunHead_kernel<<< grid, block>>>(blockHeads,
        mortonCode,
        d,
        n,
        bufLength);
}

void sahbvh_computeRunHash(KeyValuePair * compressed, 
						KeyValuePair * morton,
						uint * indices,
                        uint m,
						uint d,
						uint n,
						uint bufLength)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(bufLength, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeRunHash_kernel<<< grid, block>>>(compressed,
        morton,
        indices,
        m,
        d,
        n,
        bufLength);
}

void sahbvh_computeRunLength(uint * runLength,
							uint * runHeads,
							KeyValuePair * indices,
							uint nRuns,
							uint nPrimitives,
							uint bufLength)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(bufLength, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeRunLength_kernel<<< grid, block>>>(runLength,
        runHeads,
        indices,
        nRuns,
        nPrimitives,
        bufLength);
}

void sahbvh_compressRunHead(uint * compressed, 
							uint * runHeads,
							uint * indices,
							uint n)
							
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    compressRunHead_kernel<<< grid, block>>>(compressed,
        runHeads,
        indices,
        n);
}

void sahbvh_copyHash(KeyValuePair * dst,
					KeyValuePair * src,
					uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    copyHash_kernel<<< grid, block>>>(dst,
        src,
        n);
}

void sahbvh_decompressIndices(uint * decompressedIndices,
                    uint * compressedIndices,
					KeyValuePair * sorted,
					uint * offset,
					uint * runLength,
					uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    decompressIndices_kernel<<< grid, block>>>(decompressedIndices,
                                            compressedIndices,
                                            sorted,
                                          offset,
                                          runLength,
                                          n);
}

void sahbvh_writeSortedHash(KeyValuePair * dst,
							KeyValuePair * src,
							uint * indices,
							uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    writeSortedHash_kernel<<< grid, block>>>(dst,
							src,
							indices,
							n);
}

void sahbvh_countTreeBits(uint * nbits, 
                            KeyValuePair * morton,
                            uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    countTreeBits_kernel<<< grid, block>>>(nbits, 
                            morton,
                            n);
}

void sahbvh_computeClusterAabbs(Aabb * clusterAabbs,
            Aabb * primitiveAabbs,
            uint * compressedIndices,
            KeyValuePair * sorted,
            uint * offset,
            uint * runLength,
            uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeClusterAabbs_kernel<<< grid, block>>>(clusterAabbs, 
                            primitiveAabbs,
                            compressedIndices,
                            sorted,
                            offset,
                            runLength,
                            n);
}

}
