#include "hip/hip_runtime.h"
#include "cuReduceSum_implement.h"

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceBoxMin2F4_kernel(float4 *g_idata, float4 *g_odata, uint n)
{
    extern __shared__ float4 sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float4 myRange = make_float4(1e28f, 1e28f, 1e28f, 1.f);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        myRange.x = min(myRange.x, g_idata[i].x);
        myRange.y = min(myRange.y, g_idata[i].y);
        myRange.z = min(myRange.z, g_idata[i].z);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) {
            myRange.x = min(myRange.x, g_idata[i+blockSize].x);
            myRange.y = min(myRange.y, g_idata[i+blockSize].y);
            myRange.z = min(myRange.z, g_idata[i+blockSize].z);
        }
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = myRange;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            myRange.x = min(myRange.x, sdata[tid + 256].x);
            myRange.y = min(myRange.y, sdata[tid + 256].y);
            myRange.z = min(myRange.y, sdata[tid + 256].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            myRange.x = min(myRange.x, sdata[tid + 128].x);
            myRange.y = min(myRange.y, sdata[tid + 128].y);
            myRange.z = min(myRange.z, sdata[tid + 128].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            myRange.x = min(myRange.x, sdata[tid +  64].x); 
            myRange.y = min(myRange.y, sdata[tid + 64].y);
            myRange.z = min(myRange.z, sdata[tid + 64].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    

    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float4 * smem = sdata;
        if (blockSize >=  64) {
            myRange.x = min(myRange.x, smem[tid + 32].x);
            myRange.y = min(myRange.y, smem[tid + 32].y);
            myRange.z = min(myRange.y, smem[tid + 32].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            myRange.x = min(myRange.x, smem[tid + 16].x);
            myRange.y = min(myRange.y, smem[tid + 16].y);
            myRange.z = min(myRange.z, smem[tid + 16].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            myRange.x = min(myRange.x, smem[tid +  8].x);
            myRange.y = min(myRange.y, smem[tid +  8].y);
            myRange.z = min(myRange.z, smem[tid +  8].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            myRange.x = min(myRange.x, smem[tid +  4].x);
            myRange.y = min(myRange.y, smem[tid +  4].y);
            myRange.z = min(myRange.z, smem[tid +  4].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            myRange.x = min(myRange.x, smem[tid +  2].x);
            myRange.y = min(myRange.y, smem[tid +  2].y);
            myRange.z = min(myRange.z, smem[tid +  2].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        
        if (blockSize >=   2) { 
            myRange.x = min(myRange.x, smem[tid +  1].x);
            myRange.y = min(myRange.y, smem[tid +  1].y);
            myRange.z = min(myRange.y, smem[tid +  1].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceBoxMin1F4_kernel(Aabb *g_idata, float4 *g_odata, uint n)
{
    extern __shared__ float4 sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float4 myRange = make_float4(1e28f, 1e28f, 1e28f, 1.f);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        myRange.x = min(myRange.x, g_idata[i].low.x);
        myRange.y = min(myRange.y, g_idata[i].low.y);
        myRange.z = min(myRange.z, g_idata[i].low.z);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) {
            myRange.x = min(myRange.x, g_idata[i+blockSize].low.x);
            myRange.y = min(myRange.y, g_idata[i+blockSize].low.y);
            myRange.z = min(myRange.z, g_idata[i+blockSize].low.z);
        }
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = myRange;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            myRange.x = min(myRange.x, sdata[tid + 256].x);
            myRange.y = min(myRange.y, sdata[tid + 256].y);
            myRange.z = min(myRange.y, sdata[tid + 256].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            myRange.x = min(myRange.x, sdata[tid + 128].x);
            myRange.y = min(myRange.y, sdata[tid + 128].y);
            myRange.z = min(myRange.z, sdata[tid + 128].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            myRange.x = min(myRange.x, sdata[tid +  64].x); 
            myRange.y = min(myRange.y, sdata[tid + 64].y);
            myRange.z = min(myRange.z, sdata[tid + 64].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    

    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float4 * smem = sdata;
        if (blockSize >=  64) {
            myRange.x = min(myRange.x, smem[tid + 32].x);
            myRange.y = min(myRange.y, smem[tid + 32].y);
            myRange.z = min(myRange.y, smem[tid + 32].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            myRange.x = min(myRange.x, smem[tid + 16].x);
            myRange.y = min(myRange.y, smem[tid + 16].y);
            myRange.z = min(myRange.z, smem[tid + 16].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            myRange.x = min(myRange.x, smem[tid +  8].x);
            myRange.y = min(myRange.y, smem[tid +  8].y);
            myRange.z = min(myRange.z, smem[tid +  8].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            myRange.x = min(myRange.x, smem[tid +  4].x);
            myRange.y = min(myRange.y, smem[tid +  4].y);
            myRange.z = min(myRange.z, smem[tid +  4].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            myRange.x = min(myRange.x, smem[tid +  2].x);
            myRange.y = min(myRange.y, smem[tid +  2].y);
            myRange.z = min(myRange.z, smem[tid +  2].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        
        if (blockSize >=   2) { 
            myRange.x = min(myRange.x, smem[tid +  1].x);
            myRange.y = min(myRange.y, smem[tid +  1].y);
            myRange.z = min(myRange.y, smem[tid +  1].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceBoxMax2F4_kernel(float4 *g_idata, float4 *g_odata, uint n)
{
    extern __shared__ float4 sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float4 myRange = make_float4(-1e28f, -1e28f, -1e28f, 1.f);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        myRange.x = max(myRange.x, g_idata[i].x);
        myRange.y = max(myRange.y, g_idata[i].y);
        myRange.z = max(myRange.z, g_idata[i].z);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) {
            myRange.x = max(myRange.x, g_idata[i+blockSize].x);
            myRange.y = max(myRange.y, g_idata[i+blockSize].y);
            myRange.z = max(myRange.z, g_idata[i+blockSize].z);
        }
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = myRange;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            myRange.x = max(myRange.x, sdata[tid + 256].x);
            myRange.y = max(myRange.y, sdata[tid + 256].y);
            myRange.z = max(myRange.y, sdata[tid + 256].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            myRange.x = max(myRange.x, sdata[tid + 128].x);
            myRange.y = max(myRange.y, sdata[tid + 128].y);
            myRange.z = max(myRange.z, sdata[tid + 128].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            myRange.x = max(myRange.x, sdata[tid +  64].x); 
            myRange.y = max(myRange.y, sdata[tid + 64].y);
            myRange.z = max(myRange.z, sdata[tid + 64].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    

    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float4 * smem = sdata;
        if (blockSize >=  64) {
            myRange.x = max(myRange.x, smem[tid + 32].x);
            myRange.y = max(myRange.y, smem[tid + 32].y);
            myRange.z = max(myRange.y, smem[tid + 32].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            myRange.x = max(myRange.x, smem[tid + 16].x);
            myRange.y = max(myRange.y, smem[tid + 16].y);
            myRange.z = max(myRange.z, smem[tid + 16].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            myRange.x = max(myRange.x, smem[tid +  8].x);
            myRange.y = max(myRange.y, smem[tid +  8].y);
            myRange.z = max(myRange.z, smem[tid +  8].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            myRange.x = max(myRange.x, smem[tid +  4].x);
            myRange.y = max(myRange.y, smem[tid +  4].y);
            myRange.z = max(myRange.z, smem[tid +  4].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            myRange.x = max(myRange.x, smem[tid +  2].x);
            myRange.y = max(myRange.y, smem[tid +  2].y);
            myRange.z = max(myRange.z, smem[tid +  2].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        
        if (blockSize >=   2) { 
            myRange.x = max(myRange.x, smem[tid +  1].x);
            myRange.y = max(myRange.y, smem[tid +  1].y);
            myRange.z = max(myRange.y, smem[tid +  1].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceBoxMax1F4_kernel(Aabb *g_idata, float4 *g_odata, uint n)
{
    extern __shared__ float4 sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    float4 myRange = make_float4(-1e28f, -1e28f, -1e28f, 1.f);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        myRange.x = max(myRange.x, g_idata[i].high.x);
        myRange.y = max(myRange.y, g_idata[i].high.y);
        myRange.z = max(myRange.z, g_idata[i].high.z);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) {
            myRange.x = max(myRange.x, g_idata[i+blockSize].high.x);
            myRange.y = max(myRange.y, g_idata[i+blockSize].high.y);
            myRange.z = max(myRange.z, g_idata[i+blockSize].high.z);
        }
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = myRange;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            myRange.x = max(myRange.x, sdata[tid + 256].x);
            myRange.y = max(myRange.y, sdata[tid + 256].y);
            myRange.z = max(myRange.y, sdata[tid + 256].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            myRange.x = max(myRange.x, sdata[tid + 128].x);
            myRange.y = max(myRange.y, sdata[tid + 128].y);
            myRange.z = max(myRange.z, sdata[tid + 128].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            myRange.x = max(myRange.x, sdata[tid + 64].x); 
            myRange.y = max(myRange.y, sdata[tid + 64].y);
            myRange.z = max(myRange.z, sdata[tid + 64].z);
            sdata[tid] = myRange; 
        } 
        __syncthreads(); 
    }
    

    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float4 * smem = sdata;
        if (blockSize >=  64) {
            myRange.x = max(myRange.x, smem[tid + 32].x);
            myRange.y = max(myRange.y, smem[tid + 32].y);
            myRange.z = max(myRange.y, smem[tid + 32].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            myRange.x = max(myRange.x, smem[tid + 16].x);
            myRange.y = max(myRange.y, smem[tid + 16].y);
            myRange.z = max(myRange.z, smem[tid + 16].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            myRange.x = max(myRange.x, smem[tid +  8].x);
            myRange.y = max(myRange.y, smem[tid +  8].y);
            myRange.z = max(myRange.z, smem[tid +  8].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            myRange.x = max(myRange.x, smem[tid +  4].x);
            myRange.y = max(myRange.y, smem[tid +  4].y);
            myRange.z = max(myRange.z, smem[tid +  4].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            myRange.x = max(myRange.x, smem[tid +  2].x);
            myRange.y = max(myRange.y, smem[tid +  2].y);
            myRange.z = max(myRange.z, smem[tid +  2].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
        
        if (blockSize >=   2) { 
            myRange.x = max(myRange.x, smem[tid +  1].x);
            myRange.y = max(myRange.y, smem[tid +  1].y);
            myRange.z = max(myRange.y, smem[tid +  1].z);
            smem[tid].x = myRange.x;
            smem[tid].y = myRange.y;
            smem[tid].z = myRange.z;
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

extern "C" {
    
void cuReduce_Box_Min1_Flt4(float4 * dst, Aabb * src,
                    uint n, uint nBlocks, uint nThreads)
{
    dim3 dimBlock(nThreads, 1, 1);
    dim3 dimGrid(nBlocks, 1, 1);
	uint smemSize = (nThreads <= 32) ? 64 * 16 : nThreads * 16;
	
	if (isPow2(n)) {
		switch (nThreads)
		{
		case 512:
			reduceBoxMin1F4_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMin1F4_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMin1F4_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMin1F4_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMin1F4_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMin1F4_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMin1F4_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMin1F4_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMin1F4_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMin1F4_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
	else {
		switch (nThreads)
		{
		case 512:
			reduceBoxMin1F4_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMin1F4_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMin1F4_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMin1F4_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMin1F4_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMin1F4_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMin1F4_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMin1F4_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMin1F4_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMin1F4_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
}

void cuReduce_Box_Min2_Flt4(float4 * dst, float4 * src,
                    uint n, uint nBlocks, uint nThreads)
{
    dim3 dimBlock(nThreads, 1, 1);
    dim3 dimGrid(nBlocks, 1, 1);
	uint smemSize = (nThreads <= 32) ? 64 * 16 : nThreads * 16;
	
	if (isPow2(n)) {
		switch (nThreads)
		{
		case 512:
			reduceBoxMin2F4_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMin2F4_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMin2F4_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMin2F4_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMin2F4_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMin2F4_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMin2F4_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMin2F4_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMin2F4_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMin2F4_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
	else {
		switch (nThreads)
		{
		case 512:
			reduceBoxMin2F4_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMin2F4_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMin2F4_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMin2F4_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMin2F4_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMin2F4_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMin2F4_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMin2F4_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMin2F4_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMin2F4_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
}

void cuReduce_Box_Max1_Flt4(float4 * dst, Aabb * src,
                    uint n, uint nBlocks, uint nThreads)
{
    dim3 dimBlock(nThreads, 1, 1);
    dim3 dimGrid(nBlocks, 1, 1);
	uint smemSize = (nThreads <= 32) ? 64 * 16 : nThreads * 16;
	
	if (isPow2(n)) {
		switch (nThreads)
		{
		case 512:
			reduceBoxMax1F4_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMax1F4_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMax1F4_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMax1F4_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMax1F4_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMax1F4_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMax1F4_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMax1F4_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMax1F4_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMax1F4_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
	else {
		switch (nThreads)
		{
		case 512:
			reduceBoxMax1F4_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMax1F4_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMax1F4_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMax1F4_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMax1F4_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMax1F4_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMax1F4_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMax1F4_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMax1F4_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMax1F4_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
}

void cuReduce_Box_Max2_Flt4(float4 * dst, float4 * src,
                    uint n, uint nBlocks, uint nThreads)
{
    dim3 dimBlock(nThreads, 1, 1);
    dim3 dimGrid(nBlocks, 1, 1);
	uint smemSize = (nThreads <= 32) ? 64 * 16 : nThreads * 16;
	
	if (isPow2(n)) {
		switch (nThreads)
		{
		case 512:
			reduceBoxMax2F4_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMax2F4_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMax2F4_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMax2F4_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMax2F4_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMax2F4_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMax2F4_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMax2F4_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMax2F4_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMax2F4_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
	else {
		switch (nThreads)
		{
		case 512:
			reduceBoxMax2F4_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceBoxMax2F4_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceBoxMax2F4_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceBoxMax2F4_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceBoxMax2F4_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceBoxMax2F4_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceBoxMax2F4_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceBoxMax2F4_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceBoxMax2F4_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceBoxMax2F4_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
}

}
