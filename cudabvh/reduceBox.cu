#include "hip/hip_runtime.h"
#include "reduceBox_implement.h"

inline __device__ void resetAabb(Aabb & dst)
{
    dst.low = make_float3(10e9, 10e9, 10e9);
    dst.high = make_float3(-10e9, -10e9, -10e9);
}

inline __device__ void expandAabb(Aabb & dst, float3 p)
{
    if(p.x < dst.low.x) dst.low.x = p.x - TINY_VALUE;
    if(p.y < dst.low.y) dst.low.y = p.y - TINY_VALUE;
    if(p.z < dst.low.z) dst.low.z = p.z - TINY_VALUE;
    if(p.x > dst.high.x) dst.high.x = p.x + TINY_VALUE;
    if(p.y > dst.high.y) dst.high.y = p.y + TINY_VALUE;
    if(p.z > dst.high.z) dst.high.z = p.z + TINY_VALUE;
}

inline __device__ void expandAabb(Aabb & dst, float4 p)
{
    if(p.x < dst.low.x) dst.low.x = p.x - TINY_VALUE;
    if(p.y < dst.low.y) dst.low.y = p.y - TINY_VALUE;
    if(p.z < dst.low.z) dst.low.z = p.z - TINY_VALUE;
    if(p.x > dst.high.x) dst.high.x = p.x + TINY_VALUE;
    if(p.y > dst.high.y) dst.high.y = p.y + TINY_VALUE;
    if(p.z > dst.high.z) dst.high.z = p.z + TINY_VALUE;
}

inline __device__ void expandAabb(Aabb & dst, const Aabb & src)
{
    if(src.low.x < dst.low.x) dst.low.x = src.low.x - TINY_VALUE;
    if(src.low.y < dst.low.y) dst.low.y = src.low.y - TINY_VALUE;
    if(src.low.z < dst.low.z) dst.low.z = src.low.z - TINY_VALUE;
    if(src.high.x > dst.high.x) dst.high.x = src.high.x + TINY_VALUE;
    if(src.high.y > dst.high.y) dst.high.y = src.high.y + TINY_VALUE;
    if(src.high.z > dst.high.z) dst.high.z = src.high.z + TINY_VALUE;
}

inline __device__ void expandAabb(Aabb & dst, volatile Aabb * src)
{
    if(src->low.x < dst.low.x) dst.low.x = src->low.x - TINY_VALUE;
    if(src->low.y < dst.low.y) dst.low.y = src->low.y - TINY_VALUE;
    if(src->low.z < dst.low.z) dst.low.z = src->low.z - TINY_VALUE;
    if(src->high.x > dst.high.x) dst.high.x = src->high.x + TINY_VALUE;
    if(src->high.y > dst.high.y) dst.high.y = src->high.y + TINY_VALUE;
    if(src->high.z > dst.high.z) dst.high.z = src->high.z + TINY_VALUE;
}

inline __device__ void copyVola(volatile Aabb * dst, const Aabb & src)
{
    dst->low.x = src.low.x;
    dst->low.y = src.low.y;
    dst->low.z = src.low.z;
    dst->high.x = src.high.x;
    dst->high.y = src.high.y;
    dst->high.z = src.high.z;
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceAabbByAabb_kernel(Aabb *g_idata, Aabb *g_odata, unsigned int n)
{
    extern __shared__ Aabb sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    Aabb mySum; resetAabb(mySum);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        expandAabb(mySum, g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            expandAabb(mySum, g_idata[i+blockSize]);  
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            expandAabb(mySum, sdata[tid + 256]);
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            expandAabb(mySum, sdata[tid + 128]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            expandAabb(mySum, sdata[tid +  64]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile Aabb * smem = sdata;
        if (blockSize >=  64) {
            expandAabb(mySum, &smem[tid + 32]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            expandAabb(mySum, &smem[tid + 16]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            expandAabb(mySum, &smem[tid +  8]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            expandAabb(mySum, &smem[tid +  4]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            expandAabb(mySum, &smem[tid +  2]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   2) { 
            expandAabb(mySum, &smem[tid +  1]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceAabbByPoints_kernel(float3 *g_idata, Aabb *g_odata, unsigned int n)
{
    extern __shared__ Aabb sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    Aabb mySum; resetAabb(mySum);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        expandAabb(mySum, g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            expandAabb(mySum, g_idata[i+blockSize]);  
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            expandAabb(mySum, sdata[tid + 256]);
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            expandAabb(mySum, sdata[tid + 128]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            expandAabb(mySum, sdata[tid +  64]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile Aabb * smem = sdata;
        if (blockSize >=  64) {
            expandAabb(mySum, &smem[tid + 32]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            expandAabb(mySum, &smem[tid + 16]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            expandAabb(mySum, &smem[tid +  8]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            expandAabb(mySum, &smem[tid +  4]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            expandAabb(mySum, &smem[tid +  2]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   2) { 
            expandAabb(mySum, &smem[tid +  1]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

extern "C" void bvhReduceAabbByAabb(Aabb *dst, Aabb *src, unsigned numAabbs, unsigned numBlocks, unsigned numThreads)
{
	dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
	int smemSize = (numThreads <= 2) ? 2 * numThreads * sizeof(Aabb) : numThreads * sizeof(Aabb);
	
	if (isPow2(numAabbs)) {
		switch (numThreads)
		{
		case 512:
			reduceAabbByAabb_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 256:
			reduceAabbByAabb_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 128:
			reduceAabbByAabb_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 64:
			reduceAabbByAabb_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 32:
			reduceAabbByAabb_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 16:
			reduceAabbByAabb_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  8:
			reduceAabbByAabb_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  4:
			reduceAabbByAabb_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  2:
			reduceAabbByAabb_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  1:
			reduceAabbByAabb_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		}
	}
	else {
		switch (numThreads)
		{
		case 512:
			reduceAabbByAabb_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 256:
			reduceAabbByAabb_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 128:
			reduceAabbByAabb_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 64:
			reduceAabbByAabb_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 32:
			reduceAabbByAabb_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 16:
			reduceAabbByAabb_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  8:
			reduceAabbByAabb_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  4:
			reduceAabbByAabb_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  2:
			reduceAabbByAabb_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  1:
			reduceAabbByAabb_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		}
	}
}

extern "C" void bvhReduceAabbByPoints(Aabb *dst, float3 *src, unsigned numPoints, unsigned numBlocks, unsigned numThreads)
{
	dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
	int smemSize = (numThreads <= 2) ? 2 * numThreads * sizeof(Aabb) : numThreads * sizeof(Aabb);
	
	if (isPow2(numPoints)) {
		switch (numThreads)
		{
		case 512:
			reduceAabbByPoints_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 256:
			reduceAabbByPoints_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 128:
			reduceAabbByPoints_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 64:
			reduceAabbByPoints_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 32:
			reduceAabbByPoints_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 16:
			reduceAabbByPoints_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  8:
			reduceAabbByPoints_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  4:
			reduceAabbByPoints_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  2:
			reduceAabbByPoints_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  1:
			reduceAabbByPoints_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		}
	}
	else {
		switch (numThreads)
		{
		case 512:
			reduceAabbByPoints_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 256:
			reduceAabbByPoints_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 128:
			reduceAabbByPoints_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 64:
			reduceAabbByPoints_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 32:
			reduceAabbByPoints_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case 16:
			reduceAabbByPoints_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  8:
			reduceAabbByPoints_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  4:
			reduceAabbByPoints_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  2:
			reduceAabbByPoints_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		case  1:
			reduceAabbByPoints_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints); break;
		}
	}
}

extern "C" void getReduceBlockThread(uint & blocks, uint & threads, uint n)
{
	threads = (n < ReduceMaxThreads*2) ? nextPow2((n + 1)/ 2) : ReduceMaxThreads;
	blocks = (n + (threads * 2 - 1)) / (threads * 2);
	if(blocks > ReduceMaxBlocks) blocks = ReduceMaxBlocks;
}

extern "C" unsigned getReduceLastNThreads(unsigned n)
{
	unsigned threads, blocks;
	getReduceBlockThread(blocks, threads, n);
	
	n = blocks;
	while(n > 1) {
		getReduceBlockThread(blocks, threads, n);
		
		n = (n + (threads*2-1)) / (threads*2);
	}
	return threads;
}
