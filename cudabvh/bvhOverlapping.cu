#include "hip/hip_runtime.h"
#include "Overlapping.cuh"
#include "Overlapping2.cuh"
#include "TetrahedronSystemInterface.h"

namespace bvhoverlap {

void writeLocation(uint * dst, uint * src, uint n)
{
    int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    
    dim3 grid(nblk, 1, 1);
    
    startAsWriteLocation_kernel<<< grid, block >>>(dst, src, n);
}

void countPairsSelfCollideExclS(uint * dst, 
                                Aabb * boxes, 
                                uint numBoxes,
								int2 * internalNodeChildIndex, 
								Aabb * internalNodeAabbs, 
								Aabb * leafNodeAabbs,
								KeyValuePair * mortonCodesAndAabbIndices,
								int * exclusionIndices)
{
    int nThreads = 64;
	dim3 block(nThreads, 1, 1);
    int nblk = iDivUp(numBoxes, nThreads);
    dim3 grid(nblk, 1, 1);

	countPairsSExclS_kernel<TETRAHEDRONSYSTEM_VICINITY_LENGTH, 64> <<< grid, block>>>(dst,
                                boxes,
                                numBoxes,
								internalNodeChildIndex, 
								internalNodeAabbs, 
								leafNodeAabbs,
								mortonCodesAndAabbIndices,
								exclusionIndices);
}

void writePairCacheSelfCollideExclS(uint2 * dst, uint * locations, 
                                uint * starts, uint * counts,
                                Aabb * boxes, uint numBoxes,
								int * rootNodeIndex, 
								int2 * internalNodeChildIndex, 
								Aabb * internalNodeAabbs, 
								Aabb * leafNodeAabbs,
								KeyValuePair * mortonCodesAndAabbIndices,
								unsigned queryIdx,
								int * exclusionIndices)
{
    int nThreads = 64;
	dim3 block(nThreads, 1, 1);
    int nblk = iDivUp(numBoxes, nThreads);
    dim3 grid(nblk, 1, 1);
	
    writePairCacheSExclS_kernel<TETRAHEDRONSYSTEM_VICINITY_LENGTH, 64> <<< grid, block>>>(dst, 
                                locations,
                                starts, counts,
                                boxes,
                                numBoxes,
								internalNodeChildIndex, 
								internalNodeAabbs, 
								leafNodeAabbs,
								mortonCodesAndAabbIndices,
								queryIdx,
								exclusionIndices);
}

void countPairs(uint * dst,
                                Aabb * boxes,
                                uint numBoxes,
								int2 * internalNodeChildIndex, 
								Aabb * internalNodeAabbs, 
								Aabb * leafNodeAabbs,
								KeyValuePair * mortonCodesAndAabbIndices)
{ 
    int tpb = 64;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(numBoxes, tpb);
    
    dim3 grid(nblk, 1, 1);
    
    countPairs_kernel<64> <<< grid, block, 16320 >>>(dst,
                                boxes,
                                numBoxes,
								internalNodeChildIndex, 
								internalNodeAabbs, 
								leafNodeAabbs,
								mortonCodesAndAabbIndices);
}

void writePairCache(uint2 * dst, uint * locations, 
                                uint * starts, uint * counts,
                              Aabb * boxes, uint numBoxes,
								int2 * internalNodeChildIndex, 
								Aabb * internalNodeAabbs, 
								Aabb * leafNodeAabbs,
								KeyValuePair * mortonCodesAndAabbIndices,
								unsigned queryIdx, unsigned treeIdx)
{
    int tpb = 64;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(numBoxes, tpb);
    
    dim3 grid(nblk, 1, 1);
    
    writePairCache_kernel<64> <<< grid, block, 16320 >>>(dst, 
                                locations,
                                starts, counts,
                                boxes,
                                numBoxes,
								internalNodeChildIndex, 
								internalNodeAabbs, 
								leafNodeAabbs,
								mortonCodesAndAabbIndices,
								queryIdx, treeIdx);
}
}
